
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


static __global__ void example_cuda_kernel(int& r, int x, int y)
{

  r =  x * y + ( x * 4 - (y/2) );
}

int static_launch_kernelB(int x, int y)
{
  int r;
  example_cuda_kernel<<<1, 1>>>(r, x, y);
  return r;
}
