#include "hip/hip_runtime.h"
#include <nvbench/nvbench.cuh>
#include "matx.h"

using namespace matx;

using vec_add_types = nvbench::type_list<float, double, cuda::std::complex<float>, cuda::std::complex<double>>;

/* Vector adding benchmarks */
template <typename ValueType>
void vector_add(nvbench::state &state, nvbench::type_list<ValueType>)
{
  // Get current parameters:
  const int x_len = static_cast<int>(state.get_int64("Vector size"));

  state.add_element_count(x_len, "NumElements");
  state.add_global_memory_reads<ValueType>(2*x_len, "DataSize");
  state.add_global_memory_writes<ValueType>(x_len);  


  tensor_t<ValueType, 1> xv{{x_len}};
  tensor_t<ValueType, 1> xv2{{x_len}};
  xv.PrefetchDevice(0);
  (xv = xv + xv2).run();
  hipDeviceSynchronize();

  state.exec( 
    [&xv, &xv2](nvbench::launch &launch) {
      (xv = xv + xv2).run((hipStream_t)launch.get_stream());
    });

}

NVBENCH_BENCH_TYPES(vector_add, NVBENCH_TYPE_AXES(vec_add_types))
  .add_int64_power_of_two_axis("Vector size", nvbench::range(22, 28, 1));


using permute_types = nvbench::type_list<float, double, cuda::std::complex<float>, cuda::std::complex<double>>;
template <typename ValueType>
void permute(nvbench::state &state, nvbench::type_list<ValueType>)
{
  auto x = make_tensor<ValueType>({1000,200,6,300});
  auto y = make_tensor<ValueType>({300,1000,6,200});

  state.add_element_count(x.TotalSize(), "NumElements");
  state.add_global_memory_reads<ValueType>(x.TotalSize(), "DataSize");
  state.add_global_memory_writes<ValueType>(x.TotalSize());    

  x.PrefetchDevice(0);
  hipDeviceSynchronize();

  state.exec( 
    [&x, &y](nvbench::launch &launch) {
      (y = x.Permute({3,0,2,1})).run((hipStream_t)launch.get_stream());
    });
}


NVBENCH_BENCH_TYPES(permute, NVBENCH_TYPE_AXES(permute_types));

using random_types = nvbench::type_list<float, double, cuda::std::complex<float>, cuda::std::complex<double>>;
template <typename ValueType>
void random(nvbench::state &state, nvbench::type_list<ValueType>)
{
  auto x = make_tensor<ValueType>({1966800});
  x.PrefetchDevice(0);

  randomGenerator_t<ValueType> gen(x.TotalSize(), 0);
  
  auto y = gen.template GetTensorView<x.Rank()>(x.Shape(), NORMAL);

  state.add_element_count(x.TotalSize(), "NumElements");
  state.add_global_memory_writes<ValueType>(x.TotalSize());    

  hipDeviceSynchronize();

  state.exec( 
    [&x, &y](nvbench::launch &launch) {
      (x = y).run((hipStream_t)launch.get_stream());
    });
}

NVBENCH_BENCH_TYPES(random, NVBENCH_TYPE_AXES(random_types));

template<typename T> T factorial(int N) {
  T prod = 1;
  for(int i=2; i<=N; i++) {
    prod = prod * i;
  }
  return prod;
}

using sphericalharmonics_types = nvbench::type_list<float, double>;
template <typename ValueType>
void sphericalharmonics(nvbench::state &state, nvbench::type_list<ValueType>)
{
  int l = 5;
  int m = 4;
  int n = 600;
  ValueType dx = M_PI/n;
  
  auto col = range<0>({n+1},ValueType(0), ValueType(dx));
  auto az = range<0>({2*n+1}, ValueType(0), ValueType(dx));

  auto [phi, theta] = meshgrid(az, col);

  auto Plm = lcollapse<3>(legendre(l, m, cos(theta)));

  ValueType a = (2*l+1)*factorial<ValueType>(l-m);
  ValueType b = 4*M_PI*factorial<ValueType>(l+m);
  ValueType C = cuda::std::sqrt(a/b);

  auto Ylm = C * Plm * exp(cuda::std::complex<ValueType>(0,1)*(m*phi));
  auto [ Xm, Ym, Zm ] = sph2cart(phi, ValueType(M_PI)/2 - theta, abs(real(Ylm)));

  // Work around C++17 restriction, structured bindings cannot be captured
  auto XXm = Xm;
  auto YYm = Ym;
  auto ZZm = Zm;

  // Output location
  auto X = make_tensor<ValueType>(Xm.Shape());
  auto Y = make_tensor<ValueType>(Ym.Shape());
  auto Z = make_tensor<ValueType>(Zm.Shape());

  hipDeviceSynchronize();

  state.add_element_count(n+1, "Elements");

  state.exec( 
    [&](nvbench::launch &launch) {
      (X=XXm, Y=YYm, Z=ZZm).run((hipStream_t)launch.get_stream());
    });
}

NVBENCH_BENCH_TYPES(sphericalharmonics, NVBENCH_TYPE_AXES(sphericalharmonics_types));
