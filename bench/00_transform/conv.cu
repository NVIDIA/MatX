#include "matx.h"
#include <nvbench/nvbench.cuh>
#include "matx/core/half_complex.h"
#include "matx/core/nvtx.h"

using namespace matx;

using conv_types =
    nvbench::type_list<matxFp16Complex, cuda::std::complex<float>, cuda::std::complex<double>, float, double>;

/* FFT benchmarks */
template <typename ValueType>
void conv1d_direct_4d_batch(nvbench::state &state,
                            nvbench::type_list<ValueType>)
{
  cudaExecutor exec{0};
  auto out = make_tensor<ValueType>({4, 2, 14, 288 + 4096 + 133 - 1});
  auto at = make_tensor<ValueType>({ 4, 2, 14, 133});
  auto bt = make_tensor<ValueType>({ 4, 2, 14, 288 + 4096});

  out.PrefetchDevice(0);
  at.PrefetchDevice(0);
  bt.PrefetchDevice(0);

  exec.sync();
  MATX_NVTX_START_RANGE( "Exec", matx_nvxtLogLevels::MATX_NVTX_LOG_ALL, 1 )
  state.exec(
      [&out, &at, &bt](nvbench::launch &launch) { (out = conv1d(at, bt, MATX_C_MODE_FULL)).run(cudaExecutor(launch.get_stream())); });
  MATX_NVTX_END_RANGE( 1 )
  
}
NVBENCH_BENCH_TYPES(conv1d_direct_4d_batch, NVBENCH_TYPE_AXES(conv_types));


template <typename ValueType>
void conv1d_direct_2d_batch(nvbench::state &state,
                            nvbench::type_list<ValueType>)
{
  cudaExecutor exec{0};

  auto out = make_tensor<ValueType>({4 * 2* 14, 288 + 4096 + 133 - 1});
  auto at = make_tensor<ValueType>({ 4 * 2* 14, 133});
  auto bt = make_tensor<ValueType>({ 4 * 2* 14, 288 + 4096});

  out.PrefetchDevice(0);
  at.PrefetchDevice(0);
  bt.PrefetchDevice(0);

  exec.sync();

  state.exec(
      [&out, &at, &bt](nvbench::launch &launch) { (out = conv1d(at, bt, MATX_C_MODE_FULL)).run(cudaExecutor(launch.get_stream())); });
}
NVBENCH_BENCH_TYPES(conv1d_direct_2d_batch, NVBENCH_TYPE_AXES(conv_types));

template <typename ValueType>
void conv1d_direct_large(nvbench::state &state,
                            nvbench::type_list<ValueType>)
{
  cudaExecutor exec{0};
  auto at = make_tensor<ValueType>({state.get_int64("Signal Size")});
  auto bt = make_tensor<ValueType>({state.get_int64("Filter Size")});
  auto out = make_tensor<ValueType>({at.Size(at.Rank()-1) + bt.Size(bt.Rank()-1) - 1});

  out.PrefetchDevice(0);
  at.PrefetchDevice(0);
  bt.PrefetchDevice(0);

  (out = conv1d(at, bt, MATX_C_MODE_FULL)).run(exec);

  exec.sync();

  state.exec(
      [&out, &at, &bt](nvbench::launch &launch) { (out = conv1d(at, bt, MATX_C_MODE_FULL)).run(cudaExecutor(launch.get_stream())); });
}
NVBENCH_BENCH_TYPES(conv1d_direct_large, NVBENCH_TYPE_AXES(conv_types))
  .add_int64_power_of_two_axis("Filter Size", nvbench::range(3, 11, 1))
  .add_int64_power_of_two_axis("Signal Size", nvbench::range(12, 24, 1));

template <typename ValueType>
void conv1d_fft_large(nvbench::state &state,
                            nvbench::type_list<ValueType>)
{
  cudaExecutor exec{0};
  auto at = make_tensor<ValueType>({state.get_int64("Signal Size")});
  auto bt = make_tensor<ValueType>({state.get_int64("Filter Size")});
  auto out = make_tensor<ValueType>({at.Size(at.Rank()-1) + bt.Size(bt.Rank()-1) - 1});

  (out = conv1d(at, bt, MATX_C_MODE_FULL, MATX_C_METHOD_FFT)).run(exec);

  out.PrefetchDevice(0);
  at.PrefetchDevice(0);
  bt.PrefetchDevice(0);

  exec.sync();

  state.exec(
      [&out, &at, &bt](nvbench::launch &launch) { (out = conv1d(at, bt, MATX_C_MODE_FULL, MATX_C_METHOD_FFT)).run(cudaExecutor(launch.get_stream())); });
}
NVBENCH_BENCH_TYPES(conv1d_fft_large, NVBENCH_TYPE_AXES(conv_types))
  .add_int64_power_of_two_axis("Filter Size", nvbench::range(3, 11, 1))
  .add_int64_power_of_two_axis("Signal Size", nvbench::range(12, 24, 1));  


template <typename ValueType>
void conv2d_direct_batch(nvbench::state &state,
                            nvbench::type_list<ValueType>)
{
  cudaExecutor exec{0};
  auto at = make_tensor<ValueType>({256, 1024, 1024});
  auto bt = make_tensor<ValueType>({256, 16, 16});
  auto out = make_tensor<ValueType>({256, 
                                     at.Size(1) + bt.Size(1) - 1,
                                     at.Size(2) + bt.Size(2) - 1});

  out.PrefetchDevice(0);
  at.PrefetchDevice(0);
  bt.PrefetchDevice(0);

  exec.sync();

  state.exec(
      [&out, &at, &bt](nvbench::launch &launch) { (out = conv2d(at, bt, MATX_C_MODE_FULL)).run(cudaExecutor(launch.get_stream())); });

  auto seconds = state.get_summary("Batch GPU").get_float64("value");
  auto &flops = state.add_summary("TFLOPS");

  flops.set_string("hint", "item_rate");
  flops.set_string("short_name", "TFLOPS");
  flops.set_string("description", "Trillions of operations per second");

  if constexpr (is_complex_v<ValueType>) {
    flops.set_float64("value", static_cast<double>(2 * out.Size(2) * out.Size(1) * out.Size(0) * bt.Size(2) * bt.Size(1) * 4) / seconds / 1e12);
  } else {
    flops.set_float64("value", static_cast<double>(2 * out.Size(2) * out.Size(1) * out.Size(0) * bt.Size(2) * bt.Size(1)) / seconds / 1e12);
  }
}
NVBENCH_BENCH_TYPES(conv2d_direct_batch, NVBENCH_TYPE_AXES(conv_types));
