#include "hip/hip_runtime.h"
#include <matx.h>

using namespace matx;

int main() {

  auto A = make_tensor<float>({4, 5});
  auto MD0 = make_tensor<float>({4});
  auto AD0 = make_tensor<float>({4});

  randomGenerator_t<float> randData(A.TotalSize(), 0);
  auto randTensor1 = randData.GetTensorView<2>(shape, NORMAL);
  (A = randTensor1).run();    
  
  // Initialize max and average to 0
  (MD1 = 0).run();
  (AD1 = 0).run();

  // TODO: Reduce all rows of A by max where each reduction is a separate value in the vector MD1



  printf("A:\n");
  A.Print();
  printf("Max:\n");
  MD1.Print();
  printf("Sum:\n");
  AD1.Print();

  return 0;
}
