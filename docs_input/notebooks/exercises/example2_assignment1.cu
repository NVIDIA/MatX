#include "hip/hip_runtime.h"
#include <matx.h>

using namespace matx;

/**
 * MatX training assignment 2. This training goes through tensor operations that
 * were learned in the 02_operators notebook. Uncomment each verification block
 * as you go to ensure your solutions are correct.
 */

int main() {
  auto A = make_tensor<float>({2, 3});
  auto B = make_tensor<float>({2, 3});
  auto V = make_tensor<float>({3});

  /****************************************************************************************************
   * Initialize tensor A with increasing values from 0.5 to 3.0 in steps of 0.4,
   *and tensor V from -1 to -3 in steps of -1.
   ****************************************************************************************************/

  /*** End editing ***/

  // Verify init is correct
  float step = 0.5;
  for (int row = 0; row < A.Size(0); row++) {
    for (int col = 0; col < A.Size(1); col++) {
      if (A(row, col) != step) {
        printf("Mismatch in A init view! actual = %f, expected = %f\n",
               A(row, col), step);
        exit(-1);
      }
      step += 0.5;
    }
  }

  for (int col = 0; col < V.Size(0); col++) {
    if (V(col) != (-1 + col * -1)) {
      printf("Mismatch in A init view! actual = %f, expected = %f\n", V(col),
             (float)(-1 + col * -1));
      exit(-1);
    }
  }

  A.Print();
  V.Print();
  printf("Init verification passed!\n");

  /****************************************************************************************************
   * Add 5.0 to all elements of A and store the results back in A
   ****************************************************************************************************/

  /*** End editing ***/

  hipStreamSynchronize(0);

  step = 0.5;
  for (int row = 0; row < A.Size(0); row++) {
    for (int col = 0; col < A.Size(1); col++) {
      if (A(row, col) != (5.0 + step)) {
        printf("Mismatch in A sum view! actual = %f, expected = %f\n",
               A(row, col), 5.0 + step);
        exit(-1);
      }
      step += 0.5;
    }
  }

  A.Print();
  printf("Sum verification passed!\n");

  /****************************************************************************************************
   * Clone V to match the dimensions of A, and subtract V from A. The results
   * should be stored in A
   *
   * https://devtech-compute.gitlab-master-pages.nvidia.com/matx/quickstart.html#increasing-dimensionality
   * https://devtech-compute.gitlab-master-pages.nvidia.com/matx/api/tensorview.html#_CPPv4I0_iEN4matx12tensor_tE
   *
   ****************************************************************************************************/
  /// auto tvs = ;
  /*** End editing. ***/

  // hipStreamSynchronize(0);

  // step = 0.5;
  // for (int row = 0; row < A.Size(0); row++) {
  //   for (int col = 0; col < A.Size(1); col++) {
  //     if (A(row, col) != (5.0 + step - tvs(row, col))) {
  //       printf("Mismatch in A sub view! actual = %f, expected = %f\n", A(row,
  //       col), 5.0 + step - tvs(row, col)); exit(-1);
  //     }
  //     step += 0.5;
  //   }
  // }

  // A.Print();
  // tvs.Print();
  // printf("Clone verification passed!\n");

  /****************************************************************************************************
   * Raise the matrix A to the power of 2 and multiply the output by two. Next,
   * subtract the vector V from each row. Store the result in tensor B.
   *
   * https://devtech-compute.gitlab-master-pages.nvidia.com/matx/api/tensorops.html#_CPPv4N4matx3powE2Op2Op
   ****************************************************************************************************/

  /*** End editing ***/

  hipStreamSynchronize(0);

  for (int row = 0; row < B.Size(0); row++) {
    for (int col = 0; col < B.Size(1); col++) {
      if (B(row, col) != powf(A(row, col), 2) * 2 - V(col)) {
        printf("Mismatch in B init view! actual = %f, expected = %f\n",
               B(row, col), powf(A(row, col), 2) * 2 - V(col));
        exit(-1);
      }
    }
  }

  B.Print();
  printf("Mixed verification passed!\n");

  return 0;
}
