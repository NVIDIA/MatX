#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include <matx.h>

using namespace matx;

int main() {

  auto A = make_tensor<float>({4, 5});
  auto MD0 = make_tensor<float>();
  auto AD0 = make_tensor<float>();

  (A = random<float>({4, 5}, NORMAL)).run();    
  
  // Initialize max and average to 0
  (MD0 = 0).run();
  (AD0 = 0).run();

  // TODO: Perform a max and sum reduction of A into MD0 and AD0, respectively.


  printf("A:\n");
  print(A);
  printf("Max: %f\n", MD0());
  printf("Sum: %f\n", AD0());  

  return 0;
}
