#include <matx.h>

using namespace matx;

int main() {

  auto t2 = make_tensor<int>({5, 4});

  // Initialize the tensor linearly
  t2.SetVals({{1, 2, 3, 4},
        {5, 6, 7, 8},
        {9, 10, 11, 12},
        {13, 14, 15, 16},
        {17, 18, 19, 20}});

  t2.PrefetchDevice(0);

  // TODO: Permute the view t2 such that the two dimensions are swapped
  auto t2p = ...;

  t2p.Print();

  return 0;
}
