#include <matx.h>

using namespace matx;

/**
 * MatX training assignment 1. This training goes through basic tensor
 * operations that were learned in the 01_introduction notebook. Uncomment each
 * verification block as you go to ensure your solutions are correct.
 */

int main() {

  /****************************************************************************************************
   * Create a rank-2 tensor data object of ints with 5 rows and 4 columns called
   *"t2"
   *https://devtech-compute.gitlab-master-pages.nvidia.com/matx/quickstart.html#tensor-views
   ****************************************************************************************************/

  /*** End editing ***/

  /****************************************************************************************************
   * Initialize the t2 view to a 4x5 matrix of increasing values starting at 1
   * https://devtech-compute.gitlab-master-pages.nvidia.com/matx/quickstart.html#tensor-views
   ****************************************************************************************************/
  // t2 = ;
  /*** End editing ***/

  t2.PrefetchDevice(0);

  /****************************************************************************************************
   * Get a slice of the second and third rows with all columns
   * https://devtech-compute.gitlab-master-pages.nvidia.com/matx/quickstart.html#slicing-and-dicing
   *****************************************************************************************************/
  auto t2s = t2;
  /*** End editing ***/

  // Verify slice is correct
  // for (int row = 1; row <= 2; row++) {
  //   for (int col = 0; col < t2.Size(1); col++) {
  //     if (t2(row, col) != t2s(row - 1, col)) {
  //       printf("Mismatch in sliced view! actual = %d, expected = %d\n",
  //       t2s(row - 1, col), t2(row, col)); exit(-1);
  //     }
  //   }
  // }

  // t2s.Print();
  // printf("Slice verification passed!\n");

  /****************************************************************************************************
   * Take the slice and clone it into a 3D tensor with new outer dimensions as
   *follows: First dim: keep existing row dimension from t2s Second dim: 2 Third
   *dim: keep existing col dimension from t2s
   https://devtech-compute.gitlab-master-pages.nvidia.com/matx/quickstart.html#increasing-dimensionality
   *****************************************************************************************************/
  auto t3c = t2s;
  /*** End editing ***/

  // Verify clone
  // for (int first = 0; first < t3c.Size(0); first++) {
  //   for (int sec = 0; sec < t3c.Size(1); sec++) {
  //     for (int third = 0; third < t3c.Size(2); third++) {
  //       if (t3c(first, sec, third) != t2s(first, third)) {
  //         printf("Mismatch in cloned view! actual = %d, expected = %d\n",
  //         t3c(first, sec, third), t2s(first, third)); exit(-1);
  //       }
  //     }
  //   }
  // }

  // t3c.Print();
  // printf("Clone verification passed!\n");

  /****************************************************************************************************
   * Permute the two outer dimensions of the cloned tensor
   * https://devtech-compute.gitlab-master-pages.nvidia.com/matx/quickstart.html#permuting
   *****************************************************************************************************/
  auto t3p = t3c;
  /*** End editing ***/

  // Verify clone
  // for (int first = 0; first < t3p.Size(0); first++) {
  //   for (int sec = 0; sec < t3p.Size(1); sec++) {
  //     for (int third = 0; third < t3p.Size(2); third++) {
  //       if (t3c(first, sec, third) != t2s(first, third)) {
  //         printf("Mismatch in permuted view! actual = %d, expected = %d\n",
  //         t3c(first, sec, third), t2s(sec, third)); exit(-1);
  //       }
  //     }
  //   }
  // }

  // t3p.Print();
  // printf("Permute verification passed!\n");

  return 0;
}
