#include "hip/hip_runtime.h"
#include <matx.h>

using namespace matx;

int main() {

  using complex = cuda::std::complex<float>;

  index_t signal_size = 16;
  index_t filter_size = 3;
  index_t filtered_size = signal_size + filter_size - 1;

  // Create time domain buffers
  auto sig_time  = make_tensor<complex>({signal_size});
  auto filt_time = make_tensor<complex>({filter_size});
  auto time_out  = make_tensor<complex>({filtered_size});

  // Frequency domain buffers
  auto sig_freq  = make_tensor<complex>({filtered_size});
  auto filt_freq = make_tensor<complex>({filtered_size});

  // Fill the time domain signals with data
  for (index_t i = 0; i < signal_size; i++) {
    sig_time(b,i) = {-1.0f * (2.0f * static_cast<float>(i % 2) + 1.0f) *
                          (static_cast<float>(i % 10) / 10.0f) +
                      0.1f,
                  -1.0f * (static_cast<float>(i % 2) == 0.0f) *
                          (static_cast<float>(i % 10) / 5.0f) -
                      0.1f};
  }
  for (index_t i = 0; i < filter_size; i++) {
    filt_time(b,i) = {static_cast<float>(i) / static_cast<float>(filter_size),
                    static_cast<float>(-i) / static_cast<float>(filter_size) +
                        0.5f};
  }

  // TODO: Perform FFT convolution
  // Perform the FFT in-place on both signal and filter, do an element-wise multiply of the two, then IFFT that output


  // TODO: Perform a time-domain convolution
  

  hipStreamSynchronize(0);

  // Compare signals
  for (index_t i = 0; i < filtered_size; i++) {
      if (  fabs(time_out(i).real() - sig_freq(i).real()) > 0.001 || 
            fabs(time_out(i).imag() - sig_freq(i).imag()) > 0.001) {
          printf("Verification failed at item %lld. Direct=%f%+.2fj, FFT=%f%+.2fj\n", i,
            time_out(i).real(), time_out(i).imag(), sig_freq(i).real(), sig_freq(i).imag());
          return -1;
      }
  }

  std::cout << "Verification successful" << std::endl;

  return 0;
}
