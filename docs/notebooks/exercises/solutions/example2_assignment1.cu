#include "hip/hip_runtime.h"
#include <matx.h>

using namespace matx;

int main() {

  tensorShape_t<2> shape({2, 3});
  tensor_t<float, 2> A(shape);
  tensor_t<float, 2> B(shape);
  tensor_t<float, 1> V({3});

  /****************************************************************************************************
   * Initialize tensor A with increasing values from 0.5 to 3.0 in steps of 0.5,
   *and tensor V from -1 to -3 in steps of -1.
   ****************************************************************************************************/
  A.SetVals({{0.5, 1, 1.5}, {2.0, 2.5, 3.0}});

  V.SetVals({-1, -2, -3});
  /*** End editing ***/

  // Verify init is correct
  float step = 0.5;
  for (int row = 0; row < A.Size(0); row++) {
    for (int col = 0; col < A.Size(1); col++) {
      if (A(row, col) != step) {
        printf("Mismatch in A init view! actual = %f, expected = %f\n",
               A(row, col), step);
        exit(-1);
      }
      step += 0.5;
    }
  }

  for (int col = 0; col < V.Size(0); col++) {
    if (V(col) != (-1 + col * -1)) {
      printf("Mismatch in A init view! actual = %f, expected = %f\n", V(col),
             (float)(-1 + col * -1));
      exit(-1);
    }
  }

  A.Print();
  V.Print();
  printf("Init verification passed!\n");

  /****************************************************************************************************
   * Add 5.0 to all elements of A and store the results back in A
   ****************************************************************************************************/
  (A = A + 5.0).run();
  /*** End editing ***/

  hipStreamSynchronize(0);

  step = 0.5;
  for (int row = 0; row < A.Size(0); row++) {
    for (int col = 0; col < A.Size(1); col++) {
      if (A(row, col) != (5.0 + step)) {
        printf("Mismatch in A sum view! actual = %f, expected = %f\n",
               A(row, col), 5.0 + step);
        exit(-1);
      }
      step += 0.5;
    }
  }

  A.Print();
  printf("Sum verification passed!\n");

  /****************************************************************************************************
   * Clone V to match the dimensions of A, and subtract V from A. The results
   * should be stored in A
   *
   * https://devtech-compute.gitlab-master-pages.nvidia.com/matx/quickstart.html#increasing-dimensionality
   * https://devtech-compute.gitlab-master-pages.nvidia.com/matx/api/tensorview.html#_CPPv4I0_iEN4matx12tensor_tE
   *
   ****************************************************************************************************/
  auto tvs = V.Clone<2>({A.Size(0), matxKeepDim});
  (A = A - tvs).run();
  /*** End editing ***/

  hipStreamSynchronize(0);

  step = 0.5;
  for (int row = 0; row < A.Size(0); row++) {
    for (int col = 0; col < A.Size(1); col++) {
      if (A(row, col) != (5.0 + step - tvs(row, col))) {
        printf("Mismatch in A sub view! actual = %f, expected = %f\n",
               A(row, col), 5.0 + step - tvs(row, col));
        exit(-1);
      }
      step += 0.5;
    }
  }

  A.Print();
  tvs.Print();
  printf("Clone verification passed!\n");

  /****************************************************************************************************
   * Raise the matrix A to the power of 2 and multiply the output by two. Next,
   * subtract the vector V from each row. Store the result in tensor B.
   *
   * https://devtech-compute.gitlab-master-pages.nvidia.com/matx/api/tensorops.html#_CPPv4N4matx3powE2Op2Op
   ****************************************************************************************************/
  (B = (pow(A, 2) * 2) - V).run();
  /*** End editing ***/

  hipStreamSynchronize(0);

  for (int row = 0; row < B.Size(0); row++) {
    for (int col = 0; col < B.Size(1); col++) {
      if (B(row, col) != powf(A(row, col), 2) * 2 - V(col)) {
        printf("Mismatch in B init view! actual = %f, expected = %f\n",
               B(row, col), powf(A(row, col), 2) * 2 - V(col));
        exit(-1);
      }
    }
  }

  B.Print();
  printf("Mixed verification passed!\n");

  return 0;
}
