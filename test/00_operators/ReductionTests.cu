////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "matx_pybind.h"
#include "test_types.h"
#include "utilities.h"
#include "gtest/gtest.h"
#include <type_traits>

using namespace matx;

template <typename TensorType>
class ReductionTestsComplex : public ::testing::Test {
};
template <typename TensorType>
class ReductionTestsFloat : public ::testing::Test {
};
template <typename TensorType>
class ReductionTestsNumeric : public ::testing::Test {
};
template <typename TensorType>
class ReductionTestsNumericNonComplex : public ::testing::Test {
};
template <typename TensorType>
class ReductionTestsFloatNonComplex : public ::testing::Test {
};
template <typename TensorType>
class ReductionTestsFloatNonComplexNonHalf : public ::testing::Test {
};
template <typename TensorType>
class ReductionTestsIntegral : public ::testing::Test {
};
template <typename TensorType>
class ReductionTestsBoolean : public ::testing::Test {
};
template <typename TensorType>
class ReductionTestsFloatHalf : public ::testing::Test {
};
template <typename TensorType>
class ReductionTestsNumericNoHalf : public ::testing::Test {
};
template <typename TensorType>
class ReductionTestsAll : public ::testing::Test {
};

TYPED_TEST_SUITE(ReductionTestsAll, MatXAllTypes);
TYPED_TEST_SUITE(ReductionTestsComplex, MatXComplexTypes);
TYPED_TEST_SUITE(ReductionTestsFloat, MatXFloatTypes);
TYPED_TEST_SUITE(ReductionTestsNumeric, MatXNumericTypes);
TYPED_TEST_SUITE(ReductionTestsIntegral, MatXAllIntegralTypes);
TYPED_TEST_SUITE(ReductionTestsNumericNonComplex,
                 MatXNumericNonComplexTypes);
TYPED_TEST_SUITE(ReductionTestsFloatNonComplex, MatXFloatNonComplexTypes);
TYPED_TEST_SUITE(ReductionTestsFloatNonComplexNonHalf,
                 MatXFloatNonComplexNonHalfTypes);
TYPED_TEST_SUITE(ReductionTestsBoolean, MatXBoolTypes);
TYPED_TEST_SUITE(ReductionTestsFloatHalf, MatXFloatHalfTypes);
TYPED_TEST_SUITE(ReductionTestsNumericNoHalf, MatXNumericNoHalfTypes);


TYPED_TEST(ReductionTestsFloatNonComplexNonHalf, VarianceStd)
{
  MATX_ENTER_HANDLER();

  auto pb = std::make_unique<MatXPybind>();
  constexpr index_t size = 100;
  pb->InitAndRunTVGenerator<TypeParam>("00_operators", "stats", "run", {size});

  tensor_t<TypeParam, 0> t0;
  tensor_t<TypeParam, 1> t1({size});
  pb->NumpyToTensorView(t1, "x");

  var(t0, t1, 0);
  MATX_TEST_ASSERT_COMPARE(pb, t0, "var", 0.01);

  stdd(t0, t1, 0);
  MATX_TEST_ASSERT_COMPARE(pb, t0, "std", 0.01);

  MATX_EXIT_HANDLER();
}

TYPED_TEST(ReductionTestsFloatNonComplexNonHalf, Sum)
{
  MATX_ENTER_HANDLER();
  {
    tensor_t<TypeParam, 0> t0;

    auto t4 = ones<float>({30, 40, 50, 60});
    auto t3 = ones<float>({30, 40, 50});
    auto t2 = ones<float>({30, 40});
    auto t1 = ones<float>({30});

    sum(t0, t4, 0);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(
        t0(), (TypeParam)(t4.Size(0) * t4.Size(1) * t4.Size(2) * t4.Size(3))));

    sum(t0, t3, 0);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(
        t0(), (TypeParam)(t3.Size(0) * t3.Size(1) * t3.Size(2))));

    sum(t0, t2, 0);
    hipStreamSynchronize(0);
    EXPECT_TRUE(
        MatXUtils::MatXTypeCompare(t0(), (TypeParam)(t2.Size(0) * t2.Size(1))));

    sum(t0, t1, 0);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(t1.Size(0))));
  }
  {
    tensor_t<TypeParam, 1> t1({30});

    auto t4 = ones<float>({30, 40, 50, 60});
    auto t3 = ones<float>({30, 40, 50});
    auto t2 = ones<float>({30, 40});
    // t4.Print();
    sum(t1, t4, 0);

    hipStreamSynchronize(0);
    for (index_t i = 0; i < t1.Size(0); i++) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(
          t1(i), (TypeParam)(t4.Size(1) * t4.Size(2) * t4.Size(3))));
    }

    sum(t1, t3, 0);
    hipStreamSynchronize(0);
    for (index_t i = 0; i < t1.Size(0); i++) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(
          t1(i), (TypeParam)(t3.Size(1) * t3.Size(2))));
    }

    sum(t1, t2, 0);
    hipStreamSynchronize(0);
    for (index_t i = 0; i < t1.Size(0); i++) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (TypeParam)(t2.Size(1))));
    }
  }

  {
    tensor_t<TypeParam, 2> t2({30, 40});

    auto t4 = ones<float>({30, 40, 50, 60});
    auto t3 = ones<float>({30, 40, 50});

    sum(t2, t4, 0);
    hipStreamSynchronize(0);
    for (index_t i = 0; i < t2.Size(0); i++) {
      for (index_t j = 0; j < t2.Size(1); j++) {
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(
            t2(i, j), (TypeParam)(t4.Size(2) * t4.Size(3))));
      }
    }

    sum(t2, t3, 0);
    hipStreamSynchronize(0);
    for (index_t i = 0; i < t2.Size(0); i++) {
      for (index_t j = 0; j < t2.Size(1); j++) {
        EXPECT_TRUE(
            MatXUtils::MatXTypeCompare(t2(i, j), (TypeParam)(t3.Size(2))));
      }
    }
  }

  MATX_EXIT_HANDLER();
}

TEST(ReductionTests, Any)
{
  MATX_ENTER_HANDLER();
  using TypeParam = float;
  {
    tensor_t<TypeParam, 0> t0;

    tensor_t<float, 1> t1({30});
    tensor_t<float, 2> t2({30, 40});
    tensor_t<float, 3> t3({30, 40, 50});
    tensor_t<float, 4> t4({30, 40, 50, 60});

    (t1 = zeros<float>(t1.Shape())).run();
    (t2 = zeros<float>(t2.Shape())).run();
    (t3 = zeros<float>(t3.Shape())).run();
    (t4 = zeros<float>(t4.Shape())).run();
    hipStreamSynchronize(0);

    t1(5) = 5.0;
    t3(1, 1, 1) = 6.0;

    any(t0, t4);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(0)));

    any(t0, t3);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(1)));

    any(t0, t2);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(0)));

    any(t0, t1);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(1)));
  }

  MATX_EXIT_HANDLER();
}

TEST(ReductionTests, All)
{
  MATX_ENTER_HANDLER();
  using TypeParam = float;
  {
    tensor_t<TypeParam, 0> t0;

    tensor_t<float, 1> t1({30});
    tensor_t<float, 2> t2({30, 40});
    tensor_t<float, 3> t3({30, 40, 50});
    tensor_t<float, 4> t4({30, 40, 50, 60});

    (t1 = ones<float>(t1.Shape())).run();
    (t2 = ones<float>(t2.Shape())).run();
    (t3 = ones<float>(t3.Shape())).run();
    (t4 = ones<float>(t4.Shape())).run();
    hipStreamSynchronize(0);

    t1(5) = 0.0;
    t3(1, 1, 1) = 0.0;

    all(t0, t4);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(1)));

    all(t0, t3);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(0)));

    all(t0, t2);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(1)));

    all(t0, t1);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(0)));
  }

  MATX_EXIT_HANDLER();
}

TEST(ReductionTests, Median)
{
  MATX_ENTER_HANDLER();
  using TypeParam = float;
  {
    tensor_t<TypeParam, 0> t0{};
    tensor_t<TypeParam, 1> t1e{{10}};
    tensor_t<TypeParam, 1> t1o{{11}};
    tensor_t<TypeParam, 2> t2e{{2, 4}};
    tensor_t<TypeParam, 2> t2o{{2, 5}};
    tensor_t<TypeParam, 1> t1out{{2}};

    t1e.SetVals({1, 3, 8, 2, 9, 6, 7, 4, 5, 0});
    t1o.SetVals({1, 3, 8, 2, 9, 6, 7, 4, 5, 0, 10});
    t2e.SetVals({{2, 4, 1, 3}, {3, 1, 2, 4}});
    t2o.SetVals({{2, 4, 1, 3, 5}, {3, 1, 5, 2, 4}});

    median(t0, t1e);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(4.5f)));

    median(t0, t1o);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(5)));

    median(t1out, t2e);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1out(0), (TypeParam)(2.5f)));
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1out(1), (TypeParam)(2.5f)));

    median(t1out, t2o);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1out(0), (TypeParam)(3.0f)));
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1out(1), (TypeParam)(3.0f)));
  }

  MATX_EXIT_HANDLER();
}

TEST(ReductionTests, MinMax)
{
  MATX_ENTER_HANDLER();
  using TypeParam = float;
  {
    tensor_t<TypeParam, 0> t0{};
    tensor_t<index_t, 0> t0i{};    
    tensor_t<TypeParam, 1> t1o{{11}};
    tensor_t<TypeParam, 2> t2o{{2, 5}};
    tensor_t<TypeParam, 1> t1o_small{{2}};    
    tensor_t<index_t, 1> t1i_small{{2}};

    t1o.SetVals({1, 3, 8, 2, 9, 10, 6, 7, 4, 5, -1, 10, -1});
    t2o.SetVals({{2, 4, 1, 3, 5}, {3, 1, 5, 2, 4}});

    rmin(t0, t1o);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(-1)));

    rmax(t0, t1o);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(10)));    

    argmax(t0, t0i, t1o);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(10)));
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0i(), (TypeParam)(5)));

    argmin(t0, t0i, t1o);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(-1)));
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0i(), (TypeParam)(10)));    

    argmax(t1o_small, t1i_small, t2o);
    hipStreamSynchronize(0);

    // We need to convert the absolute index into relative before comparing
    auto rel = t2o.GetIdxFromAbs(t1i_small(0));
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2o(rel), (TypeParam)(5)));
    rel = t2o.GetIdxFromAbs(t1i_small(1));
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2o(rel), (TypeParam)(5)));

    argmin(t1o_small, t1i_small, t2o);
    hipStreamSynchronize(0);
    
    rel = t2o.GetIdxFromAbs(t1i_small(0));
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2o(rel), (TypeParam)(1)));
    rel = t2o.GetIdxFromAbs(t1i_small(1));
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2o(rel), (TypeParam)(1)));  
  }

  MATX_EXIT_HANDLER();
}

TEST(ReductionTests, Mean)
{
  MATX_ENTER_HANDLER();
  using TypeParam = float;
  {
    tensor_t<TypeParam, 0> t0;

    auto t4 = ones<float>({30, 40, 50, 60});
    auto t3 = ones<float>({30, 40, 50});
    auto t2 = ones<float>({30, 40});
    auto t1 = ones<float>({30});

    mean(t0, t4);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(1)));

    mean(t0, t3);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(1)));

    mean(t0, t2);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(1)));

    mean(t0, t1);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(1)));
  }
  {
    tensor_t<TypeParam, 1> t1({30});

    auto t4 = ones<float>({30, 40, 50, 60});
    auto t3 = ones<float>({30, 40, 50});
    auto t2 = ones<float>({30, 40});

    mean(t1, t4);
    hipStreamSynchronize(0);
    for (index_t i = 0; i < t1.Size(0); i++) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (TypeParam)(1)));
    }

    mean(t1, t3);
    hipStreamSynchronize(0);
    for (index_t i = 0; i < t1.Size(0); i++) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (TypeParam)(1)));
    }

    mean(t1, t2);
    hipStreamSynchronize(0);
    for (index_t i = 0; i < t1.Size(0); i++) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (TypeParam)(1)));
    }
  }

  {
    tensor_t<TypeParam, 2> t2({30, 40});

    auto t4 = ones<float>({30, 40, 50, 60});
    auto t3 = ones<float>({30, 40, 50});

    mean(t2, t4);
    hipStreamSynchronize(0);
    for (index_t i = 0; i < t2.Size(0); i++) {
      for (index_t j = 0; j < t2.Size(1); j++) {
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2(i, j), (TypeParam)(1)));
      }
    }

    mean(t2, t3);
    hipStreamSynchronize(0);
    for (index_t i = 0; i < t2.Size(0); i++) {
      for (index_t j = 0; j < t2.Size(1); j++) {
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2(i, j), (TypeParam)(1)));
      }
    }
  }

  MATX_EXIT_HANDLER();
}

TYPED_TEST(ReductionTestsNumericNonComplex, Prod)
{
  MATX_ENTER_HANDLER();
  {
    tensor_t<TypeParam, 0> t0;

    std::array<index_t, 2> s2{3, 4};
    std::array<index_t, 1> s1{3};

    tensor_t<TypeParam, 1> t1{s1};
    tensor_t<TypeParam, 2> t2{s2};
    TypeParam t1p = (TypeParam)1;
    for (int i = 0; i < t1.Size(0); i++) {
      t1(i) = static_cast<value_promote_t<TypeParam>>((float)rand() /
                                                      (float)INT_MAX * 2.0f);
      t1p *= t1(i);
    }

    TypeParam t2p = (TypeParam)1;
    for (int i = 0; i < t2.Size(0); i++) {
      for (int j = 0; j < t2.Size(1); j++) {
        t2(i, j) = static_cast<value_promote_t<TypeParam>>(
            (float)rand() / (float)INT_MAX * 2.0f);
        t2p *= t2(i, j);
      }
    }

    prod(t0, t2);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), t2p));

    prod(t0, t1);
    hipStreamSynchronize(0);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), t1p));
  }

  MATX_EXIT_HANDLER();
}

// TYPED_TEST(ReductionTestsNumericNonComplex, Reduce)
// {
//   MATX_ENTER_HANDLER();
//   {
//     tensor_t<TypeParam, 0> t0data;
//     tensor_t<TypeParam, 4> t4data({30, 40, 50, 60});

//     auto t0 = t0data.View();
//     auto t4 = t4data.View();
//     for(index_t i = 0 ; i < t4.Size(0); i++) {
//       for(index_t j = 0 ; j < t4.Size(1); j++) {
//         for(index_t k = 0 ; k < t4.Size(2); k++) {
//           for(index_t l = 0 ; l < t4.Size(3); l++) {
//             t4(i,j,k,l) = (TypeParam) (i + j + k + l - 20);
//           }
//         }
//       }
//     }

//     reduce(t0, t4, reduceOpMax<TypeParam>(), 0);
//     hipStreamSynchronize(0);
//     EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam) (t4.Size(0) +
//     t4.Size(1) + t4.Size(2) + t4.Size(3) - 20 - 4) ));

//     reduce(t0, t4, reduceOpMin<TypeParam>(), 0);
//     hipStreamSynchronize(0);
//     EXPECT_TRUE(MatXUtils::MatXTypeCompare(t0(), (TypeParam)(-20) ));
//   }

//   MATX_EXIT_HANDLER();
// }
