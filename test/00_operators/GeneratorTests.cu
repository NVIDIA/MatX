#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "matx_pybind.h"
#include "test_types.h"
#include "utilities.h"
#include "gtest/gtest.h"
#include <type_traits>

using namespace matx;

template <typename TensorType>
class BasicGeneratorTestsComplex : public ::testing::Test {
};
template <typename TensorType>
class BasicGeneratorTestsFloat : public ::testing::Test {
};
template <typename TensorType>
class BasicGeneratorTestsNumeric : public ::testing::Test {
};
template <typename TensorType>
class BasicGeneratorTestsNumericNonComplex : public ::testing::Test {
};
template <typename TensorType>
class BasicGeneratorTestsFloatNonComplex : public ::testing::Test {
};
template <typename TensorType>
class BasicGeneratorTestsFloatNonComplexNonHalf : public ::testing::Test {
};
template <typename TensorType>
class BasicGeneratorTestsIntegral : public ::testing::Test {
};
template <typename TensorType>
class BasicGeneratorTestsBoolean : public ::testing::Test {
};
template <typename TensorType>
class BasicGeneratorTestsFloatHalf : public ::testing::Test {
};
template <typename TensorType>
class BasicGeneratorTestsNumericNoHalf : public ::testing::Test {
};
template <typename TensorType>
class BasicGeneratorTestsAll : public ::testing::Test {
};

TYPED_TEST_SUITE(BasicGeneratorTestsAll, MatXAllTypes);
TYPED_TEST_SUITE(BasicGeneratorTestsComplex, MatXComplexTypes);
TYPED_TEST_SUITE(BasicGeneratorTestsFloat, MatXFloatTypes);
TYPED_TEST_SUITE(BasicGeneratorTestsNumeric, MatXNumericTypes);
TYPED_TEST_SUITE(BasicGeneratorTestsIntegral, MatXAllIntegralTypes);
TYPED_TEST_SUITE(BasicGeneratorTestsNumericNonComplex,
                 MatXNumericNonComplexTypes);
TYPED_TEST_SUITE(BasicGeneratorTestsFloatNonComplex, MatXFloatNonComplexTypes);
TYPED_TEST_SUITE(BasicGeneratorTestsFloatNonComplexNonHalf,
                 MatXFloatNonComplexNonHalfTypes);
TYPED_TEST_SUITE(BasicGeneratorTestsBoolean, MatXBoolTypes);
TYPED_TEST_SUITE(BasicGeneratorTestsFloatHalf, MatXFloatHalfTypes);
TYPED_TEST_SUITE(BasicGeneratorTestsNumericNoHalf, MatXNumericNoHalfTypes);

TYPED_TEST(BasicGeneratorTestsFloatNonComplex, Windows)
{
  MATX_ENTER_HANDLER();

  auto pb = std::make_unique<MatXPybind>();
  const index_t win_size = 100;
  pb->InitAndRunTVGenerator<TypeParam>("00_operators", "window", "run",
                                       {win_size});
  std::array<index_t, 1> shape({win_size});
  auto ov = make_tensor<TypeParam>(shape);

  (ov = hanning_x(shape)).run();
  MATX_TEST_ASSERT_COMPARE(pb, ov, "hanning", 0.01);

  (ov = hamming_x(shape)).run();
  MATX_TEST_ASSERT_COMPARE(pb, ov, "hamming", 0.01);

  (ov = bartlett_x(shape)).run();
  MATX_TEST_ASSERT_COMPARE(pb, ov, "bartlett", 0.01);

  (ov = blackman_x(shape)).run();
  MATX_TEST_ASSERT_COMPARE(pb, ov, "blackman", 0.01);
  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsAll, Diag)
{
  MATX_ENTER_HANDLER();
  {
    tensor_t<TypeParam, 2> tc({10, 10});
    tensor_t<TypeParam, 1> td({10});

    for (int i = 0; i < 10; i++) {
      for (int j = 0; j < 10; j++) {

        // The half precision headers define competing constructors for
        // double/float, so we need to cast
        TypeParam val(static_cast<value_promote_t<TypeParam>>(i * 10 + j));
        tc(i, j) = val;
      }
    }

    (td = diag(tc)).run();
    hipStreamSynchronize(0);

    for (int i = 0; i < 10; i++) {
      for (int j = 0; j < 10; j++) {
        if (i == j) {
          MATX_ASSERT_EQ(td(i), tc(i, j));
        }
      }
    }
  }
  MATX_EXIT_HANDLER();
}

TEST(OperatorTests, Kron)
{
  MATX_ENTER_HANDLER();
  using dtype = int;
  auto pb = std::make_unique<MatXPybind>();
  pb->InitTVGenerator<dtype>("00_operators", "kron_operator", {});
  pb->RunTVGenerator("run");

  tensor_t<dtype, 2> bv({2, 2});
  tensor_t<dtype, 2> ov({8, 8});
  bv.SetVals({{1, -1}, {-1, 1}});

  (ov = kron(eye({4, 4}), bv)).run();
  hipStreamSynchronize(0);
  MATX_TEST_ASSERT_COMPARE(pb, ov, "square", 0);

  tensor_t<dtype, 2> av({2, 3});
  tensor_t<dtype, 2> ov2({4, 6});
  av.SetVals({{1, 2, 3}, {4, 5, 6}});

  (ov2 = kron(av, ones({2, 2}))).run();
  hipStreamSynchronize(0);
  MATX_TEST_ASSERT_COMPARE(pb, ov2, "rect", 0);

  MATX_EXIT_HANDLER();
}

TEST(OperatorTests, MeshGrid)
{
  MATX_ENTER_HANDLER();
  using dtype = int;
  auto pb = std::make_unique<MatXPybind>();
  constexpr index_t xd = 3;
  constexpr index_t yd = 5;
  pb->InitAndRunTVGenerator<dtype>("00_operators", "meshgrid_operator", "run",
                                   {yd, xd});

  tensor_t<dtype, 2> xv({yd, xd});
  tensor_t<dtype, 2> yv({yd, xd});

  (xv = meshgrid_x({1, xd, xd}, {1, yd, yd})).run();
  (yv = meshgrid_y({1, xd, xd}, {1, yd, yd})).run();
  hipStreamSynchronize(0);
  MATX_TEST_ASSERT_COMPARE(pb, xv, "X", 0);
  MATX_TEST_ASSERT_COMPARE(pb, yv, "Y", 0);

  MATX_EXIT_HANDLER();
}



TYPED_TEST(BasicGeneratorTestsAll, Zeros)
{
  MATX_ENTER_HANDLER();
  index_t count = 100;
  std::array<index_t, 1> s({count});

  auto t1 = make_tensor<TypeParam>(s);

  (t1 = zeros(s)).run();
  hipStreamSynchronize(0);

  for (index_t i = 0; i < count; i++) {
    if constexpr (IsHalfType<TypeParam>()) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (float)0));
    }
    else {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (TypeParam)0));
    }
  }
  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsAll, Ones)
{
  MATX_ENTER_HANDLER();
  index_t count = 100;
  std::array<index_t, 1> s({count});
  auto t1 = make_tensor<TypeParam>(s);

  (t1 = ones(s)).run();
  hipStreamSynchronize(0);

  for (index_t i = 0; i < count; i++) {
    if constexpr (IsHalfType<TypeParam>()) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (float)1));
    }
    else {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (TypeParam)1));
    }
  }
  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsNumericNonComplex, Range)
{
  MATX_ENTER_HANDLER();
  index_t count = 100;
  tensor_t<TypeParam, 1> t1{{count}};

  (t1 = range_x(t1.Shape(), 1, 1)).run();
  hipStreamSynchronize(0);

  TypeParam one = 1;
  TypeParam two = 1;
  TypeParam three = 1;

  for (index_t i = 0; i < count; i++) {
    TypeParam it = static_cast<value_promote_t<TypeParam>>(i);
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), it + one));
  }

  {
    (t1 = t1 * t1).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count; i++) {
      TypeParam it = static_cast<value_promote_t<TypeParam>>(i);
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (it + one) * (it + one)));
    }
  }

  {
    (t1 = t1 * two).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count; i++) {
      TypeParam it = static_cast<value_promote_t<TypeParam>>(i);
      EXPECT_TRUE(
          MatXUtils::MatXTypeCompare(t1(i), ((it + one) * (it + one)) * two));
    }
  }

  {
    (t1 = three * t1).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count; i++) {
      TypeParam it = static_cast<value_promote_t<TypeParam>>(i);
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), ((it + one) * (it + one)) *
                                                        two * three));
    }
  }

  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsNumericNonComplex, Linspace)
{
  MATX_ENTER_HANDLER();
  index_t count = 100;
  tensor_t<TypeParam, 1> t1{{count}};
  auto s = t1.Shape();
  (t1 = linspace_x(s, (TypeParam)1, (TypeParam)100)).run();
  hipStreamSynchronize(0);

  for (index_t i = 0; i < count; i++) {
    EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), i + 1));
  }

  {
    (t1 = t1 + t1).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count; i++) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (i + 1) + (i + 1)));
    }
  }

  {
    (t1 = (TypeParam)1 + t1).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count; i++) {
      EXPECT_TRUE(
          MatXUtils::MatXTypeCompare(t1(i), (i + 1.0f) + (i + 1.0f) + 1.0f));
    }
  }

  {
    (t1 = t1 + (TypeParam)2).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count; i++) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), (i + 1) + (i + 1) + 1 + 2));
    }
  }

  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsFloatNonComplex, Logspace)
{
  MATX_ENTER_HANDLER();
  index_t count = 20;
  tensor_t<TypeParam, 1> t1{{count}};
  TypeParam start = 1.0f;
  TypeParam stop = 2.0f;
  auto s = t1.Shape();
  (t1 = logspace_x(s, start, stop)).run();

  hipStreamSynchronize(0);

  // Use doubles for verification since half operators have no equivalent host
  // types
  double step = (static_cast<double>(stop) - static_cast<double>(start)) /
                static_cast<double>(s[s.size() - 1] - 1);

  for (index_t i = 0; i < count; i++) {
    if constexpr (IsHalfType<TypeParam>()) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(
          t1(i),
          cuda::std::powf(10, static_cast<double>(start) +
                                  static_cast<double>(step) *
                                      static_cast<double>(i)),
          2));
    }
    else {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(
          t1(i),
          cuda::std::powf(10, static_cast<double>(start) +
                                  static_cast<double>(step) *
                                      static_cast<double>(i)),
          0.01));
    }
  }

  MATX_EXIT_HANDLER();
}


TYPED_TEST(BasicGeneratorTestsNumeric, Eye)
{
  MATX_ENTER_HANDLER();
  index_t count = 100;

  tensor_t<TypeParam, 1> t1({count});
  tensor_t<TypeParam, 2> t2({count, count});
  tensor_t<TypeParam, 3> t3({count, count, count});
  tensor_t<TypeParam, 4> t4({count, count, count, count});

  t1.PrefetchDevice(0);
  t2.PrefetchDevice(0);
  t3.PrefetchDevice(0);
  t4.PrefetchDevice(0);

  auto eye1 = eye<TypeParam>({count});
  auto eye2 = eye<TypeParam>({count, count});
  auto eye3 = eye<TypeParam>({count, count, count});
  auto eye4 = eye<TypeParam>({count, count, count, count});

  (t1 = eye1).run();
  (t2 = eye2).run();
  (t3 = eye3).run();
  (t4 = eye4).run();

  t1.PrefetchHost(0);
  t2.PrefetchHost(0);
  t3.PrefetchHost(0);
  t4.PrefetchHost(0);

  TypeParam one = 1.0f;
  TypeParam zero = 0.0f;

  hipDeviceSynchronize();

  for (index_t i = 0; i < count; i++) {
    if (i == 0)
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), one));
    else
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), zero));
  }

  for (index_t i = 0; i < count; i++) {
    for (index_t j = 0; j < count; j++) {
      if (i == j)
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2(i, j), one));
      else
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2(i, j), zero));
    }
  }

  for (index_t i = 0; i < count; i++) {
    for (index_t j = 0; j < count; j++) {
      for (index_t k = 0; k < count; k++) {
        if (i == j && j == k)
          EXPECT_TRUE(MatXUtils::MatXTypeCompare(t3(i, j, k), one));
        else
          EXPECT_TRUE(MatXUtils::MatXTypeCompare(t3(i, j, k), zero));
      }
    }
  }

  for (index_t i = 0; i < count; i++) {
    for (index_t j = 0; j < count; j++) {
      for (index_t k = 0; k < count; k++) {
        for (index_t l = 0; l < count; l++) {
          if (i == j && j == k && k == l)
            EXPECT_TRUE(MatXUtils::MatXTypeCompare(t4(i, j, k, l), one));
          else
            EXPECT_TRUE(MatXUtils::MatXTypeCompare(t4(i, j, k, l), zero));
        }
      }
    }
  }
  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsNumeric, Diag)
{
  MATX_ENTER_HANDLER();
  index_t count = 100;
  TypeParam c = GenerateData<TypeParam>();

  tensor_t<TypeParam, 1> t1({count});
  tensor_t<TypeParam, 2> t2({count, count});
  tensor_t<TypeParam, 3> t3({count, count, count});
  tensor_t<TypeParam, 4> t4({count, count, count, count});

  t1.PrefetchDevice(0);
  t2.PrefetchDevice(0);
  t3.PrefetchDevice(0);
  t4.PrefetchDevice(0);

  auto diag1 = diag<TypeParam>({count}, c);
  auto diag2 = diag<TypeParam>({count, count}, c);
  auto diag3 = diag<TypeParam>({count, count, count}, c);
  auto diag4 = diag<TypeParam>({count, count, count, count}, c);

  (t1 = diag1).run();
  (t2 = diag2).run();
  (t3 = diag3).run();
  (t4 = diag4).run();

  t1.PrefetchHost(0);
  t2.PrefetchHost(0);
  t3.PrefetchHost(0);
  t4.PrefetchHost(0);

  TypeParam zero = 0.0f;

  hipDeviceSynchronize();

  for (index_t i = 0; i < count; i++) {
    if (i == 0)
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), c));
    else
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t1(i), zero));
  }

  for (index_t i = 0; i < count; i++) {
    for (index_t j = 0; j < count; j++) {
      if (i == j)
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2(i, j), c));
      else
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2(i, j), zero));
    }
  }

  for (index_t i = 0; i < count; i++) {
    for (index_t j = 0; j < count; j++) {
      for (index_t k = 0; k < count; k++) {
        if (i == j && j == k)
          EXPECT_TRUE(MatXUtils::MatXTypeCompare(t3(i, j, k), c));
        else
          EXPECT_TRUE(MatXUtils::MatXTypeCompare(t3(i, j, k), zero));
      }
    }
  }

  for (index_t i = 0; i < count; i++) {
    for (index_t j = 0; j < count; j++) {
      for (index_t k = 0; k < count; k++) {
        for (index_t l = 0; l < count; l++) {
          if (i == j && j == k && k == l)
            EXPECT_TRUE(MatXUtils::MatXTypeCompare(t4(i, j, k, l), c));
          else
            EXPECT_TRUE(MatXUtils::MatXTypeCompare(t4(i, j, k, l), zero));
        }
      }
    }
  }
  MATX_EXIT_HANDLER();
}


TYPED_TEST(BasicGeneratorTestsComplex, HermitianTranspose)
{
  MATX_ENTER_HANDLER();
  index_t count0 = 100;
  index_t count1 = 200;
  tensor_t<TypeParam, 2> t2({count0, count1});
  tensor_t<TypeParam, 2> t2s({count1, count0});
  for (index_t i = 0; i < count0; i++) {
    for (index_t j = 0; j < count1; j++) {
      TypeParam tmp = {(float)i, (float)-j};
      t2(i, j) = tmp;
    }
  }

  (t2s = hermitianT(t2)).run();
  hipStreamSynchronize(0);

  for (index_t i = 0; i < count0; i++) {
    for (index_t j = 0; j < count1; j++) {
      EXPECT_TRUE(
          MatXUtils::MatXTypeCompare(static_cast<double>(t2s(j, i).real()),
                                     static_cast<double>(t2(i, j).real())));
      EXPECT_TRUE(
          MatXUtils::MatXTypeCompare(-static_cast<double>(t2s(j, i).imag()),
                                     static_cast<double>(t2(i, j).imag())));
    }
  }
  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsComplex, PlanarTransform)
{
  MATX_ENTER_HANDLER();
  index_t m = 10;
  index_t k = 20;
  tensor_t<TypeParam, 2> t2({m, k});
  tensor_t<typename TypeParam::value_type, 2> t2p({m * 2, k});
  for (index_t i = 0; i < m; i++) {
    for (index_t j = 0; j < k; j++) {
      TypeParam tmp = {(float)i, (float)-j};
      t2(i, j) = tmp;
    }
  }

  (t2p = planar(t2)).run();
  hipStreamSynchronize(0);

  for (index_t i = 0; i < m; i++) {
    for (index_t j = 0; j < k; j++) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2(i, j).real(), t2p(i, j)));
      EXPECT_TRUE(
          MatXUtils::MatXTypeCompare(t2(i, j).imag(), t2p(i + t2.Size(0), j)));
    }
  }
  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsComplex, InterleavedTransform)
{
  MATX_ENTER_HANDLER();
  index_t m = 10;
  index_t k = 20;
  tensor_t<TypeParam, 2> t2({m, k});
  tensor_t<typename TypeParam::value_type, 2> t2p({m * 2, k});
  for (index_t i = 0; i < 2 * m; i++) {
    for (index_t j = 0; j < k; j++) {
      if (i >= m) {
        t2p(i, j) = 2.0f;
      }
      else {
        t2p(i, j) = -1.0f;
      }
    }
  }

  (t2 = interleaved(t2p)).run();
  hipStreamSynchronize(0);

  for (index_t i = 0; i < m; i++) {
    for (index_t j = 0; j < k; j++) {
      EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2(i, j).real(), t2p(i, j)));
      EXPECT_TRUE(
          MatXUtils::MatXTypeCompare(t2(i, j).imag(), t2p(i + t2.Size(0), j)));
    }
  }
  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsAll, RepMat)
{
  MATX_ENTER_HANDLER();
  index_t count0 = 4;
  index_t count1 = 4;
  index_t same_reps = 10;
  tensor_t<TypeParam, 2> t2({count0, count1});
  tensor_t<TypeParam, 2> t2s({count0 * same_reps, count1 * same_reps});

  for (index_t i = 0; i < count0; i++) {
    for (index_t j = 0; j < count1; j++) {
      t2(i, j) = static_cast<value_promote_t<TypeParam>>(i);
    }
  }

  auto repop = repmat(t2, same_reps);
  ASSERT_TRUE(repop.Size(0) == same_reps * t2.Size(0));
  ASSERT_TRUE(repop.Size(1) == same_reps * t2.Size(1));

  (t2s = repop).run();
  hipStreamSynchronize(0);

  for (index_t i = 0; i < count0 * same_reps; i++) {
    for (index_t j = 0; j < count1 * same_reps; j++) {
      EXPECT_TRUE(
          MatXUtils::MatXTypeCompare(t2s(i, j), t2(i % count0, j % count1)));
    }
  }

  // Now a rectangular repmat
  tensor_t<TypeParam, 2> t2r({count0 * same_reps, count1 * same_reps * 2});

  auto rrepop = repmat(t2, {same_reps, same_reps * 2});
  ASSERT_TRUE(rrepop.Size(0) == same_reps * t2.Size(0));
  ASSERT_TRUE(rrepop.Size(1) == same_reps * 2 * t2.Size(1));

  (t2r = rrepop).run();
  hipStreamSynchronize(0);

  for (index_t i = 0; i < count0 * same_reps; i++) {
    for (index_t j = 0; j < count1 * same_reps * 2; j++) {
      EXPECT_TRUE(
          MatXUtils::MatXTypeCompare(t2r(i, j), t2(i % count0, j % count1)));
    }
  }
  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsNumeric, Shift)
{
  MATX_ENTER_HANDLER();
  index_t count0 = 100;
  index_t count1 = 201;
  tensor_t<TypeParam, 2> t2({count0, count1});
  tensor_t<TypeParam, 2> t2s({count0, count1});
  tensor_t<TypeParam, 2> t2s2({count0, count1});

  for (index_t i = 0; i < count0; i++) {
    for (index_t j = 0; j < count1; j++) {
      t2(i, j) = static_cast<value_promote_t<TypeParam>>(i * count1 + j);
    }
  }

  {
    (t2s = shift0(t2, 5)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(
            MatXUtils::MatXTypeCompare(t2s(i, j), t2((i + 5) % count0, j)));
      }
    }
  }

  {
    (t2s = shift1(t2, 5)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(
            MatXUtils::MatXTypeCompare(t2s(i, j), t2(i, (j + 5) % count1)));
      }
    }
  }

  {
    (t2s = shift0(shift1(t2, 5), 6)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(
            t2s(i, j), t2((i + 6) % count0, (j + 5) % count1)));
      }
    }
  }

  {
    (t2s = fftshift2D(t2)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(
            t2s(i, j), t2((i + (count0 + 1) / 2) % count0,
                          (j + (count1 + 1) / 2) % count1)));
      }
    }
  }

  {
    (t2s = ifftshift2D(t2)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(
            t2s(i, j),
            t2((i + (count0) / 2) % count0, (j + (count1) / 2) % count1)));
      }
    }
  }

  // Negative shifts
  {
    (t2s = shift0(t2, -5)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        index_t idim = i < 5 ? (t2.Size(0) - 5 + i) : (i - 5);
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2s(i, j), t2(idim, j)));
      }
    }
  }

  {
    (t2s = shift1(t2, -5)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        index_t jdim = j < 5 ? (t2.Size(1) - 5 + j) : (j - 5);
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2s(i, j), t2(i, jdim)));
      }
    }
  }

  // Large shifts
  {
    (t2s = shift0(t2, t2.Size(0) * 4)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2s(i, j), t2(i, j)));
      }
    }
  }

  {
    // Shift 4 times the size back, minus one. This should be equivalent to
    // simply shifting by -1
    (t2s = shift0(t2, -t2.Size(0) * 4 - 1)).run();
    (t2s2 = shift0(t2, -1)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(t2s(i, j), t2s2(i, j)));
      }
    }
  }

  MATX_EXIT_HANDLER();
}

TYPED_TEST(BasicGeneratorTestsNumeric, Reverse)
{
  MATX_ENTER_HANDLER();
  index_t count0 = 100;
  index_t count1 = 200;
  tensor_t<TypeParam, 2> t2({count0, count1});
  tensor_t<TypeParam, 2> t2r({count0, count1});

  for (index_t i = 0; i < count0; i++) {
    for (index_t j = 0; j < count1; j++) {
      t2(i, j) = static_cast<value_promote_t<TypeParam>>(i * count1 + j);
    }
  }

  {
    (t2r = reverseY(t2)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(
            MatXUtils::MatXTypeCompare(t2r(i, j), t2(count0 - i - 1, j)));
      }
    }
  }

  {
    (t2r = reverseX(t2)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(
            MatXUtils::MatXTypeCompare(t2r(i, j), t2(i, count1 - j - 1)));
      }
    }
  }

  {
    (t2r = reverseX(reverseY(t2))).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(MatXUtils::MatXTypeCompare(
            t2r(i, j), t2(count0 - i - 1, count1 - j - 1)));
      }
    }
  }

  // Flip versions
  {
    (t2r = flipud(t2)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(
            MatXUtils::MatXTypeCompare(t2r(i, j), t2(count0 - i - 1, j)));
      }
    }
  }

  {
    (t2r = fliplr(t2)).run();
    hipStreamSynchronize(0);

    for (index_t i = 0; i < count0; i++) {
      for (index_t j = 0; j < count1; j++) {
        EXPECT_TRUE(
            MatXUtils::MatXTypeCompare(t2r(i, j), t2(i, count1 - j - 1)));
      }
    }
  }

  MATX_EXIT_HANDLER();
}

