////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "test_types.h"
#include "utilities.h"
#include "gtest/gtest.h"

using namespace matx;

/* NOTE: CUTLASS tests are disabled for now. The compile times are too long at
 * the moment */
template <typename T> class MatMulTest : public ::testing::Test {
protected:
  void SetUp() override
  {
    CheckTestTensorCoreTypeSupport<T>();

    pb = std::make_unique<detail::MatXPybind>(); // Half precision needs a bit more
                                         // tolerance when compared to fp32
    if constexpr (is_complex_half_v<T> || is_matx_half_v<T>) {
      thresh = 0.5f;
    }
  }

  void TearDown() { pb.reset(); }

  std::unique_ptr<detail::MatXPybind> pb;
  float thresh = 0.01f;
};

template <typename TensorType>
class MatMulTestFloatTypes : public MatMulTest<TensorType> {
};

TYPED_TEST_SUITE(MatMulTestFloatTypes, MatXFloatTypes);

TYPED_TEST(MatMulTestFloatTypes, SmallRect)
{
  MATX_ENTER_HANDLER();
  constexpr index_t m = 4;
  constexpr index_t k = 8;
  constexpr index_t n = 16;
  tensor_t<TypeParam, 2> a{{m, k}};
  tensor_t<TypeParam, 2> b{{k, n}};
  tensor_t<TypeParam, 2> c{{m, n}};

  this->pb->template InitAndRunTVGenerator<TypeParam>(
      "00_transforms", "matmul_operators", "run", {m, k, n});

  this->pb->NumpyToTensorView(a, "a");
  this->pb->NumpyToTensorView(b, "b");

  matmul<decltype(c), decltype(a), decltype(b), PROVIDER_TYPE_CUBLASLT>(c, a, b);
  MATX_TEST_ASSERT_COMPARE(this->pb, c, "c", this->thresh);

  MATX_EXIT_HANDLER();
}

TYPED_TEST(MatMulTestFloatTypes, SmallRectATranspose)
{
  MATX_ENTER_HANDLER();
  constexpr index_t m = 4;
  constexpr index_t k = 8;
  constexpr index_t n = 16;
  tensor_t<TypeParam, 2> a{{k, m}};
  tensor_t<TypeParam, 2> b{{k, n}};
  tensor_t<TypeParam, 2> c{{m, n}};

  this->pb->template InitAndRunTVGenerator<TypeParam>(
      "00_transforms", "matmul_operators", "run_a_transpose", {m, k, n});

  this->pb->NumpyToTensorView(a, "a");
  this->pb->NumpyToTensorView(b, "b");

  auto at = a.PermuteMatrix();
  matmul(c, at, b);
  MATX_TEST_ASSERT_COMPARE(this->pb, c, "c", this->thresh);

  MATX_EXIT_HANDLER();
}

TYPED_TEST(MatMulTestFloatTypes, SmallRectBTranspose)
{
  MATX_ENTER_HANDLER();
  constexpr index_t m = 4;
  constexpr index_t k = 8;
  constexpr index_t n = 16;
  tensor_t<TypeParam, 2> a{{m, k}};
  tensor_t<TypeParam, 2> b{{n, k}};
  tensor_t<TypeParam, 2> c{{m, n}};

  this->pb->template InitAndRunTVGenerator<TypeParam>(
      "00_transforms", "matmul_operators", "run_b_transpose", {m, k, n});

  this->pb->NumpyToTensorView(a, "a");
  this->pb->NumpyToTensorView(b, "b");

  auto bt = b.PermuteMatrix();
  matmul(c, a, bt);
  MATX_TEST_ASSERT_COMPARE(this->pb, c, "c", this->thresh);

  MATX_EXIT_HANDLER();
}

TYPED_TEST(MatMulTestFloatTypes, SmallRectUserPointer)
{
  MATX_ENTER_HANDLER();
  constexpr index_t m = 4;
  constexpr index_t k = 8;
  constexpr index_t n = 16;
  TypeParam *ap, *bp, *cp;
  hipMallocManaged(&ap, m*k*sizeof(TypeParam));
  hipMallocManaged(&bp, k*n*sizeof(TypeParam));
  hipMallocManaged(&cp, m*n*sizeof(TypeParam));

  auto a = make_tensor<TypeParam, 2>(ap, {m, k},false);
  auto b = make_tensor<TypeParam, 2>(bp, {k, n},false);
  auto c = make_tensor<TypeParam, 2>(cp, {m, n},false);

  this->pb->template InitAndRunTVGenerator<TypeParam>(
      "00_transforms", "matmul_operators", "run", {m, k, n});

  this->pb->NumpyToTensorView(a, "a");
  this->pb->NumpyToTensorView(b, "b");

  matmul<decltype(c), decltype(a), decltype(b), PROVIDER_TYPE_CUBLASLT>(c, a, b);
  MATX_TEST_ASSERT_COMPARE(this->pb, c, "c", this->thresh);

  hipFree(ap);
  hipFree(bp);
  hipFree(cp);

  MATX_EXIT_HANDLER();
}


TYPED_TEST(MatMulTestFloatTypes, DISABLED_SmallRectTranspose)
{
  MATX_ENTER_HANDLER();
  constexpr index_t m = 4;
  constexpr index_t k = 8;
  constexpr index_t n = 16;
  tensor_t<TypeParam, 2> a{{m, k}};
  tensor_t<TypeParam, 2> b{{k, n}};
  tensor_t<TypeParam, 2> c{{m, n}};

  auto at = a.Permute({1,0});
  auto bt = b.Permute({1,0});
  auto ct = c.Permute({1,0});

  this->pb->template InitAndRunTVGenerator<TypeParam>(
      "00_transforms", "matmul_operators", "run_transpose", {m, k, n});

  this->pb->NumpyToTensorView(a, "a");
  this->pb->NumpyToTensorView(b, "b");

  matmul<decltype(ct), decltype(bt), decltype(at), PROVIDER_TYPE_CUBLASLT>(ct, bt, at);

  MATX_TEST_ASSERT_COMPARE(this->pb, ct, "c", 0.01);
  MATX_EXIT_HANDLER();
}

TYPED_TEST(MatMulTestFloatTypes, SmallSquare)
{
  MATX_ENTER_HANDLER();
  constexpr index_t m = 4;
  constexpr index_t k = 4;
  constexpr index_t n = 4;
  tensor_t<TypeParam, 2> a{{m, k}};
  tensor_t<TypeParam, 2> b{{k, n}};
  tensor_t<TypeParam, 2> c{{m, n}};

  this->pb->template InitAndRunTVGenerator<TypeParam>(
      "00_transforms", "matmul_operators", "run", {m, k, n});

  this->pb->NumpyToTensorView(a, "a");
  this->pb->NumpyToTensorView(b, "b");

  matmul<decltype(c), decltype(a), decltype(b), PROVIDER_TYPE_CUBLASLT>(c, a, b);
  MATX_TEST_ASSERT_COMPARE(this->pb, c, "c", this->thresh);

  // matmul<TypeParam, TypeParam, TypeParam, 2, PROVIDER_TYPE_CUTLASS>(c, a,
  //                                                                    b);
  // MATX_TEST_ASSERT_COMPARE(this->pb, c, "c", this->thresh);
  MATX_EXIT_HANDLER();
}

TYPED_TEST(MatMulTestFloatTypes, MediumRect)
{
  MATX_ENTER_HANDLER();
  constexpr index_t m = 128;
  constexpr index_t k = 256;
  constexpr index_t n = 512;
  tensor_t<TypeParam, 2> a{{m, k}};
  tensor_t<TypeParam, 2> b{{k, n}};
  tensor_t<TypeParam, 2> c{{m, n}};

  this->pb->template InitAndRunTVGenerator<TypeParam>(
      "00_transforms", "matmul_operators", "run", {m, k, n});

  this->pb->NumpyToTensorView(a, "a");
  this->pb->NumpyToTensorView(b, "b");

  matmul<decltype(c), decltype(a), decltype(b), PROVIDER_TYPE_CUBLASLT>(c, a, b);
  MATX_TEST_ASSERT_COMPARE(this->pb, c, "c", this->thresh);

  // matmul<TypeParam, TypeParam, TypeParam, 2, PROVIDER_TYPE_CUTLASS>(c, a,
  //                                                                    b);
  // MATX_TEST_ASSERT_COMPARE(this->pb, c, "c", this->thresh);

  MATX_EXIT_HANDLER();
}

TYPED_TEST(MatMulTestFloatTypes, MediumRectBatched)
{
  MATX_ENTER_HANDLER();
  constexpr index_t batches = 5;
  constexpr index_t m = 128;
  constexpr index_t k = 256;
  constexpr index_t n = 512;
  
  tensor_t<TypeParam, 3> a{{batches, m, k}};
  tensor_t<TypeParam, 3> b{{batches, k, n}};
  tensor_t<TypeParam, 3> c{{batches, m, n}};  

  this->pb->template InitAndRunTVGenerator<TypeParam>(
      "00_transforms", "matmul_operators", "run", {m, k, n, batches});

  this->pb->NumpyToTensorView(a, "a");
  this->pb->NumpyToTensorView(b, "b");

  matmul<decltype(c), decltype(a), decltype(b), PROVIDER_TYPE_CUBLASLT>(c, a, b);

  MATX_TEST_ASSERT_COMPARE(this->pb, c, "c", this->thresh);

  MATX_EXIT_HANDLER();
}
