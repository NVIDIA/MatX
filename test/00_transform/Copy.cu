////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "test_types.h"
#include "utilities.h"
#include "gtest/gtest.h"

using namespace matx;

template <typename TensorType>
class CopyTestsAll : public ::testing::Test
{
};

TYPED_TEST_SUITE(CopyTestsAll, MatXAllTypesAllExecs);

TYPED_TEST(CopyTestsAll, CopyOutParam)
{
  MATX_ENTER_HANDLER();

  using TestType = cuda::std::tuple_element_t<0, TypeParam>;
  using ExecType = cuda::std::tuple_element_t<1, TypeParam>;

  ExecType exec{};

  const int SZ = 5;
  TestType DEFAULT, TEST_VAL;
  if constexpr (std::is_same_v<TestType, bool>) {
    DEFAULT = true;
    TEST_VAL = false;
  } else {
    DEFAULT = {2};
    TEST_VAL = {7};
  }

  // The following tests create an N dimensional tensor, N > 0, and populates
  // the full tensor with value DEFAULT, except for the index given by
  // {SZ/2, SZ/2, ..., SZ/2}, which is TEST_VAL. This tensor is then copied
  // and we verify that the copy has the same pattern.
  #define TEST_NUM_DIMS(N) \
    do { \
      cuda::std::array<index_t, N> dims; \
      dims.fill(SZ); \
      auto in = make_tensor<TestType>(dims); \
      auto out = make_tensor<TestType>(dims); \
      (in = DEFAULT).run(exec); \
      exec.sync(); \
      cuda::std::array<index_t, N> inds; \
      inds.fill(SZ/2); \
      in(inds) = TEST_VAL; \
      exec.sync(); \
      matx::copy(out, in, exec); \
      exec.sync(); \
      ASSERT_EQ(in(inds), out(inds)); \
      ASSERT_EQ(out(inds), TEST_VAL); \
      inds.fill(0); \
      ASSERT_EQ(in(inds), out(inds)); \
      ASSERT_EQ(out(inds), DEFAULT); \
    } while (0);

  TEST_NUM_DIMS(1);
  TEST_NUM_DIMS(2);
  TEST_NUM_DIMS(3);
  TEST_NUM_DIMS(4);
  TEST_NUM_DIMS(5);
  #undef TEST_NUM_DIMS

  // 0D tensors are an exception to the above test because as scalars they can
  // only hold a single value. Thus, we create a 0D tensor with value TEST_VAL
  // and verify that the copy receives the same value.
  {
    auto in = make_tensor<TestType>({});
    auto out = make_tensor<TestType>({});
    in() = TEST_VAL;
    exec.sync();
    matx::copy(out, in, exec);
    exec.sync();
    ASSERT_EQ(in(), out());
    ASSERT_EQ(out(), TEST_VAL);
  }

  // Also test that deep copying from a slice works as expected
  {
    auto in = make_tensor<TestType>({SZ, SZ, SZ});
    auto out = make_tensor<TestType>({SZ});
    (in = DEFAULT).run(exec);
    exec.sync();
    in(0, SZ/2, 0) = TEST_VAL;
    exec.sync();
    matx::copy(out, slice<1>(in, {0,0,0}, {matxDropDim,matxEnd,matxDropDim}), exec);
    exec.sync();
    ASSERT_EQ(out.Rank(), 1);
    ASSERT_EQ(out.Size(0), SZ);
    ASSERT_EQ(out(SZ/2), TEST_VAL);
    ASSERT_EQ(out(0), DEFAULT);
  }

  if constexpr (std::is_same_v<ExecType,cudaExecutor>) {
    ASSERT_EQ(hipGetLastError(), hipSuccess);
  }

  MATX_EXIT_HANDLER();
}

TYPED_TEST(CopyTestsAll, CopyReturn)
{
  MATX_ENTER_HANDLER();

  using TestType = cuda::std::tuple_element_t<0, TypeParam>;
  using ExecType = cuda::std::tuple_element_t<1, TypeParam>;

  ExecType exec{};

  const int SZ = 5;
  TestType DEFAULT, TEST_VAL;
  if constexpr (std::is_same_v<TestType, bool>) {
    DEFAULT = true;
    TEST_VAL = false;
  } else {
    DEFAULT = {2};
    TEST_VAL = {7};
  }

  // The following tests create an N dimensional tensor, N > 0, and populates
  // the full tensor with value DEFAULT, except for the index given by
  // {SZ/2, SZ/2, ..., SZ/2}, which is TEST_VAL. This tensor is then copied
  // and we verify that the copy has the same pattern.
  #define TEST_NUM_DIMS(N) \
    do { \
      cuda::std::array<index_t, N> dims; \
      dims.fill(SZ); \
      auto in = make_tensor<TestType>(dims); \
      (in = DEFAULT).run(exec); \
      exec.sync(); \
      cuda::std::array<index_t, N> inds; \
      inds.fill(SZ/2); \
      in(inds) = TEST_VAL; \
      exec.sync(); \
      auto out = matx::copy(in, exec); \
      exec.sync(); \
      ASSERT_EQ(in(inds), out(inds)); \
      ASSERT_EQ(out(inds), TEST_VAL); \
      inds.fill(0); \
      ASSERT_EQ(in(inds), out(inds)); \
      ASSERT_EQ(out(inds), DEFAULT); \
    } while (0);

  TEST_NUM_DIMS(1);
  TEST_NUM_DIMS(2);
  TEST_NUM_DIMS(3);
  TEST_NUM_DIMS(4);
  TEST_NUM_DIMS(5);
  #undef TEST_NUM_DIMS

  // 0D tensors are an exception to the above test because as scalars they can
  // only hold a single value. Thus, we create a 0D tensor with value TEST_VAL
  // and verify that the copy receives the same value.
  {
    auto in = make_tensor<TestType>({});
    in() = TEST_VAL;
    exec.sync();
    auto out = matx::copy(in, exec);
    exec.sync();
    ASSERT_EQ(in(), out());
    ASSERT_EQ(out(), TEST_VAL);
  }

  // Also test that deep copying from a slice works as expected
  {
    auto in = make_tensor<TestType>({SZ, SZ, SZ});
    (in = DEFAULT).run(exec);
    exec.sync();
    in(0, SZ/2, 0) = TEST_VAL;
    exec.sync();
    auto out = matx::copy(slice<1>(in, {0,0,0}, {matxDropDim,matxEnd,matxDropDim}), exec);
    exec.sync();
    ASSERT_EQ(out.Rank(), 1);
    ASSERT_EQ(out.Size(0), SZ);
    ASSERT_EQ(out(SZ/2), TEST_VAL);
    ASSERT_EQ(out(0), DEFAULT);
  }

  if constexpr (std::is_same_v<ExecType,cudaExecutor>) {
    ASSERT_EQ(hipGetLastError(), hipSuccess);
  }

  MATX_EXIT_HANDLER();
}