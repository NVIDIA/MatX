////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "test_types.h"
#include "utilities.h"
#include "gtest/gtest.h"
#include "matx_pybind.h"
#include "matx_einsum.h"

using namespace matx;

template <typename TensorType> struct ContractionTestsData {
  tensor_t<TensorType, 0> t0{};
  tensor_t<TensorType, 1> t1{{10}};
  tensor_t<TensorType, 2> t2{{20, 10}};
  tensor_t<TensorType, 3> t3{{30, 20, 10}};
  tensor_t<TensorType, 4> t4{{40, 30, 20, 10}};

  tensor_t<TensorType, 2> t2s = t2.Permute({1, 0});
  tensor_t<TensorType, 3> t3s = t3.Permute({2, 1, 0});
  tensor_t<TensorType, 4> t4s = t4.Permute({3, 2, 1, 0});
};

template <typename TensorType>
class ContractionTestsComplex : public ::testing::Test,
                                public ContractionTestsData<TensorType> {
};
template <typename TensorType>
class ContractionTestsFloat : public ::testing::Test,
                              public ContractionTestsData<TensorType> {
};
template <typename TensorType>
class ContractionTestsFloatNonComplex
    : public ::testing::Test,
      public ContractionTestsData<TensorType> {
};
template <typename TensorType>
class ContractionTestsNumeric : public ::testing::Test,
                                public ContractionTestsData<TensorType> {
};
template <typename TensorType>
class ContractionTestsNumericNonComplex
    : public ::testing::Test,
      public ContractionTestsData<TensorType> {
};
template <typename TensorType>
class ContractionTestsIntegral : public ::testing::Test,
                                 public ContractionTestsData<TensorType> {
};
template <typename TensorType>
class ContractionTestsBoolean : public ::testing::Test,
                                public ContractionTestsData<TensorType> {
};
template <typename TensorType>
class ContractionTestsAll : public ::testing::Test,
                            public ContractionTestsData<TensorType> {
};

TYPED_TEST_SUITE(ContractionTestsAll, MatXAllTypes);
TYPED_TEST_SUITE(ContractionTestsComplex, MatXComplexTypes);
TYPED_TEST_SUITE(ContractionTestsFloat, MatXFloatTypes);
TYPED_TEST_SUITE(ContractionTestsFloatNonComplex, MatXFloatNonComplexTypes);
TYPED_TEST_SUITE(ContractionTestsNumeric, MatXNumericTypes);
TYPED_TEST_SUITE(ContractionTestsIntegral, MatXAllIntegralTypes);
TYPED_TEST_SUITE(ContractionTestsNumericNonComplex, MatXNumericNonComplexTypes);
TYPED_TEST_SUITE(ContractionTestsBoolean, MatXBoolTypes);

#ifdef ENABLE_CUTENSOR
TEST(ContractionTests, BasicRealFloat)
{
  MATX_ENTER_HANDLER();
  auto pb = std::make_unique<detail::MatXPybind>();
  pb->template InitAndRunTVGenerator<float>(
      "00_operators", "contraction", "run", {});  

  auto a1 = make_tensor<float>({60});
  auto b1 = make_tensor<float>({24});
  auto c2 = make_tensor<float>({5,2});

  (a1 = linspace<0>(a1.Shape(), 0.0f, static_cast<float>(a1.Size(0) - 1))).run();
  (b1 = linspace<0>(b1.Shape(), 0.0f, static_cast<float>(b1.Size(0) - 1))).run();
  auto a = a1.View({3,4,5});
  auto b = b1.View({4,3,2});

  cutensor::einsum(c2, "ijk,jil->kl", 0, a, b);
  hipStreamSynchronize(0);
  MATX_TEST_ASSERT_COMPARE(pb, c2, "c_float3d", 0.01);

  MATX_EXIT_HANDLER();
}
#endif
