#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "test_types.h"
#include "utilities.h"
#include "gtest/gtest.h"

using namespace matx;

template <typename TensorType> struct ViewTestsData {
  tensor_t<TensorType, 0> t0{};
  tensor_t<TensorType, 1> t1{{10}};
  tensor_t<TensorType, 2> t2{{20, 10}};
  tensor_t<TensorType, 3> t3{{30, 20, 10}};
  tensor_t<TensorType, 4> t4{{40, 30, 20, 10}};

  tensor_t<TensorType, 2> t2s = t2.Permute({1, 0});
  tensor_t<TensorType, 3> t3s = t3.Permute({2, 1, 0});
  tensor_t<TensorType, 4> t4s = t4.Permute({3, 2, 1, 0});
};

template <typename TensorType>
class ViewTestsComplex : public ::testing::Test,
                                public ViewTestsData<TensorType> {
};
template <typename TensorType>
class ViewTestsFloat : public ::testing::Test,
                              public ViewTestsData<TensorType> {
};
template <typename TensorType>
class ViewTestsFloatNonComplex
    : public ::testing::Test,
      public ViewTestsData<TensorType> {
};
template <typename TensorType>
class ViewTestsNumeric : public ::testing::Test,
                                public ViewTestsData<TensorType> {
};
template <typename TensorType>
class ViewTestsNumericNonComplex
    : public ::testing::Test,
      public ViewTestsData<TensorType> {
};
template <typename TensorType>
class ViewTestsIntegral : public ::testing::Test,
                                 public ViewTestsData<TensorType> {
};
template <typename TensorType>
class ViewTestsBoolean : public ::testing::Test,
                                public ViewTestsData<TensorType> {
};
template <typename TensorType>
class ViewTestsAll : public ::testing::Test,
                            public ViewTestsData<TensorType> {
};

template <typename TensorType>
class ViewTestsFloatNonComplexNonHalf : public ::testing::Test,
                            public ViewTestsData<TensorType> {
};



TYPED_TEST_SUITE(ViewTestsAll, MatXAllTypes);
TYPED_TEST_SUITE(ViewTestsComplex, MatXComplexTypes);
TYPED_TEST_SUITE(ViewTestsFloat, MatXFloatTypes);
TYPED_TEST_SUITE(ViewTestsFloatNonComplex, MatXFloatNonComplexTypes);
TYPED_TEST_SUITE(ViewTestsNumeric, MatXNumericTypes);
TYPED_TEST_SUITE(ViewTestsIntegral, MatXAllIntegralTypes);
TYPED_TEST_SUITE(ViewTestsNumericNonComplex, MatXNumericNonComplexTypes);
TYPED_TEST_SUITE(ViewTestsBoolean, MatXBoolTypes);
TYPED_TEST_SUITE(ViewTestsFloatNonComplexNonHalf, MatXFloatNonComplexNonHalfTypes);


TYPED_TEST(ViewTestsNumericNonComplex, OverlapView)
{
  MATX_ENTER_HANDLER();

  tensor_t<TypeParam, 1> a{{10}};
  a.SetVals({0, 1, 2, 3, 4, 5, 6, 7, 8, 9});
  auto ao = a.OverlapView({4}, {2});

  tensor_t<TypeParam, 2> b{{4, 4}};
  b.SetVals({{0, 1, 2, 3}, {2, 3, 4, 5}, {4, 5, 6, 7}, {6, 7, 8, 9}});
  for (index_t i = 0; i < b.Size(0); i++) {
    for (index_t j = 0; j < b.Size(1); j++) {
      ASSERT_EQ(ao(i, j), b(i, j));
    }
  }

  auto ao2 = a.OverlapView({4}, {1});

  tensor_t<TypeParam, 2> b2{{7, 4}};
  b2.SetVals({{0, 1, 2, 3},
              {1, 2, 3, 4},
              {2, 3, 4, 5},
              {3, 4, 5, 6},
              {4, 5, 6, 7},
              {5, 6, 7, 8},
              {6, 7, 8, 9}});
  for (index_t i = 0; i < b2.Size(0); i++) {
    for (index_t j = 0; j < b2.Size(1); j++) {
      ASSERT_EQ(ao2(i, j), b2(i, j));
    }
  }

  auto ao3 = a.OverlapView({4}, {3});
  tensor_t<TypeParam, 2> b3{{3, 4}};
  b3.SetVals({{0, 1, 2, 3}, {3, 4, 5, 6}, {6, 7, 8, 9}});
  for (index_t i = 0; i < b3.Size(0); i++) {
    for (index_t j = 0; j < b3.Size(1); j++) {
      ASSERT_EQ(ao3(i, j), b3(i, j));
    }
  }

  auto ao4 = a.OverlapView({3}, {2});
  tensor_t<TypeParam, 2> b4{{4, 3}};
  b4.SetVals({{0, 1, 2}, {2, 3, 4}, {4, 5, 6}, {6, 7, 8}});
  for (index_t i = 0; i < b4.Size(0); i++) {
    for (index_t j = 0; j < b4.Size(1); j++) {
      ASSERT_EQ(ao4(i, j), b4(i, j));
    }
  }

  MATX_EXIT_HANDLER();
}

TYPED_TEST(ViewTestsAll, Stride)
{
  MATX_ENTER_HANDLER();

  ASSERT_EQ(this->t1.Stride(0), 1);
  ASSERT_EQ(this->t2.Stride(1), 1);
  ASSERT_EQ(this->t3.Stride(2), 1);
  ASSERT_EQ(this->t4.Stride(3), 1);

  ASSERT_EQ(this->t2.Stride(0), this->t2.Size(1));
  ASSERT_EQ(this->t3.Stride(1), this->t3.Size(2));
  ASSERT_EQ(this->t4.Stride(2), this->t4.Size(3));

  ASSERT_EQ(this->t3.Stride(0), this->t3.Size(2) * this->t3.Size(1));
  ASSERT_EQ(this->t4.Stride(1), this->t4.Size(3) * this->t4.Size(2));

  ASSERT_EQ(this->t4.Stride(0),
            this->t4.Size(3) * this->t4.Size(2) * this->t4.Size(1));

  MATX_EXIT_HANDLER();
}


TYPED_TEST(ViewTestsIntegral, SliceStride)
{
  MATX_ENTER_HANDLER();
  this->t1.SetVals({10, 20, 30, 40, 50, 60, 70, 80, 90, 100});
  auto t1t = this->t1.Slice({0}, {matxEnd}, {2});

  for (index_t i = 0; i < this->t1.Size(0); i += 2) {
    ASSERT_EQ(this->t1(i), t1t(i / 2));
  }

  auto t1t2 = this->t1.Slice({2}, {matxEnd}, {2});

  for (index_t i = 0; i < t1t2.Size(0); i++) {
    ASSERT_EQ(30 + 20 * i, t1t2(i));
  }

  MATX_EXIT_HANDLER();
}

TYPED_TEST(ViewTestsIntegral, Slice)
{
  MATX_ENTER_HANDLER();
  auto t2t = this->t2.Slice({1, 2}, {3, 5});
  auto t3t = this->t3.Slice({1, 2, 3}, {3, 5, 7});
  auto t4t = this->t4.Slice({1, 2, 3, 4}, {3, 5, 7, 9});

#ifndef NDEBUG
  // Negative slice test
  try {
    auto t2e = this->t2.Slice({1, 2}, {1, 2});
    ASSERT_EQ(true, false);
  }
  catch (...) {
    ASSERT_EQ(true, true);
  }
#endif  

  ASSERT_EQ(t2t.Size(0), 2);
  ASSERT_EQ(t2t.Size(1), 3);

  ASSERT_EQ(t3t.Size(0), 2);
  ASSERT_EQ(t3t.Size(1), 3);
  ASSERT_EQ(t3t.Size(2), 4);

  ASSERT_EQ(t4t.Size(0), 2);
  ASSERT_EQ(t4t.Size(1), 3);
  ASSERT_EQ(t4t.Size(2), 4);
  ASSERT_EQ(t4t.Size(3), 5);

  for (index_t i = 0; i < t2t.Size(0); i++) {
    for (index_t j = 0; j < t2t.Size(1); j++) {
      ASSERT_EQ(t2t(i, j), this->t2(i + 1, j + 2));
    }
  }

  for (index_t i = 0; i < t3t.Size(0); i++) {
    for (index_t j = 0; j < t3t.Size(1); j++) {
      for (index_t k = 0; k < t3t.Size(2); k++) {
        ASSERT_EQ(t3t(i, j, k), this->t3(i + 1, j + 2, k + 3));
      }
    }
  }

  for (index_t i = 0; i < t4t.Size(0); i++) {
    for (index_t j = 0; j < t4t.Size(1); j++) {
      for (index_t k = 0; k < t4t.Size(2); k++) {
        for (index_t l = 0; l < t4t.Size(3); l++) {
          ASSERT_EQ(t4t(i, j, k, l), this->t4(i + 1, j + 2, k + 3, l + 4));
        }
      }
    }
  }
  MATX_EXIT_HANDLER();
}

TYPED_TEST(ViewTestsAll, SliceAndReduce)
{
  tensor_t<float, 2> t2t{{20, 10}};
  tensor_t<float, 3> t3t{{30, 20, 10}};

  MATX_ENTER_HANDLER();
  {
    index_t j = 0;
    auto t2sly = t2t.Slice<1>({0, j}, {matxEnd, matxDropDim});
    for (index_t i = 0; i < t2sly.Size(0); i++) {
      ASSERT_EQ(t2sly(i), t2t(i, j));
    }
  }

  {
    index_t i = 0;
    auto t2slx = t2t.Slice<1>({i, 0}, {matxDropDim, matxEnd});
    for (index_t j = 0; j < t2slx.Size(0); j++) {
      ASSERT_EQ(t2slx(j), t2t(i, j));
    }
  }

  {
    index_t j = 0;
    index_t k = 0;
    auto t3slz = t3t.Slice<1>({0, j, k}, {matxEnd, matxDropDim, matxDropDim});
    for (index_t i = 0; i < t3slz.Size(0); i++) {
      ASSERT_EQ(t3slz(i), t3t(i, j, k));
    }
  }

  {
    index_t i = 0;
    index_t k = 0;
    auto t3sly = t3t.Slice<1>({i, 0, k}, {matxDropDim, matxEnd, matxDropDim});
    for (index_t j = 0; j < t3sly.Size(0); j++) {
      ASSERT_EQ(t3sly(j), t3t(i, j, k));
    }
  }

  {
    index_t i = 0;
    index_t j = 0;
    auto t3slx = t3t.Slice<1>({i, j, 0}, {matxDropDim, matxDropDim, matxEnd});
    for (index_t k = 0; k < t3slx.Size(0); k++) {
      ASSERT_EQ(t3slx(k), t3t(i, j, k));
    }
  }

  {
    index_t k = 0;
    auto t3slzy = t3t.Slice<2>({0, 0, k}, {matxEnd, matxEnd, matxDropDim});
    for (index_t i = 0; i < t3slzy.Size(0); i++) {
      for (index_t j = 0; j < t3slzy.Size(1); j++) {
        ASSERT_EQ(t3slzy(i, j), t3t(i, j, k));
      }
    }
  }

  {
    index_t j = 0;
    auto t3slzx = t3t.Slice<2>({0, j, 0}, {matxEnd, matxDropDim, matxEnd});
    for (index_t i = 0; i < t3slzx.Size(0); i++) {
      for (index_t k = 0; k < t3slzx.Size(1); k++) {
        ASSERT_EQ(t3slzx(i, k), t3t(i, j, k));
      }
    }
  }

  {
    index_t i = 0;
    auto t3slyx = t3t.Slice<2>({i, 0, 0}, {matxDropDim, matxEnd, matxEnd});
    for (index_t j = 0; j < t3slyx.Size(0); j++) {
      for (index_t k = 0; k < t3slyx.Size(1); k++) {
        ASSERT_EQ(t3slyx(j, k), t3t(i, j, k));
      }
    }
  }
  MATX_EXIT_HANDLER();
}

TEST(BasicTensorTest, Clone)
{
  tensor_t<float, 0> t0{};
  tensor_t<float, 1> t1{{10}};
  tensor_t<float, 2> t2{{20, 10}};
  tensor_t<float, 3> t3{{30, 20, 10}};

  MATX_ENTER_HANDLER();
  // clone t0 across 0/1/2/3 dim
  auto t0c1 = t0.Clone<1>({5});
  ASSERT_EQ(t0c1.Size(0), 5);
  for (index_t i = 0; i < t0c1.Size(0); i++) {
    ASSERT_EQ(t0c1(i), t0());
  }

  auto t0c2 = t0.Clone<2>({5, 6});
  ASSERT_EQ(t0c2.Size(0), 5);
  ASSERT_EQ(t0c2.Size(1), 6);
  for (index_t i = 0; i < t0c2.Size(0); i++) {
    for (index_t j = 0; j < t0c2.Size(1); j++) {
      ASSERT_EQ(t0c2(i, j), t0());
    }
  }

  auto t0c3 = t0.Clone<3>({5, 6, 7});
  ASSERT_EQ(t0c3.Size(0), 5);
  ASSERT_EQ(t0c3.Size(1), 6);
  ASSERT_EQ(t0c3.Size(2), 7);
  for (index_t i = 0; i < t0c3.Size(0); i++) {
    for (index_t j = 0; j < t0c3.Size(1); j++) {
      for (index_t k = 0; k < t0c3.Size(2); k++) {
        ASSERT_EQ(t0c3(i, j, k), t0());
      }
    }
  }

  auto t0c4 = t0.Clone<4>({5, 6, 7, 8});
  ASSERT_EQ(t0c4.Size(0), 5);
  ASSERT_EQ(t0c4.Size(1), 6);
  ASSERT_EQ(t0c4.Size(2), 7);
  ASSERT_EQ(t0c4.Size(3), 8);
  for (index_t i = 0; i < t0c4.Size(0); i++) {
    for (index_t j = 0; j < t0c4.Size(1); j++) {
      for (index_t k = 0; k < t0c4.Size(2); k++) {
        for (index_t l = 0; l < t0c4.Size(3); l++) {
          ASSERT_EQ(t0c4(i, j, k, l), t0());
        }
      }
    }
  }

  auto t1c1 = t1.Clone<2>({5, matxKeepDim});
  ASSERT_EQ(t1c1.Size(0), 5);
  for (index_t i = 0; i < t1c1.Size(0); i++) {
    for (index_t j = 0; j < t1c1.Size(1); j++) {
      ASSERT_EQ(t1c1(i, j), t1(j));
    }
  }

  auto t1c2 = t1.Clone<3>({5, 6, matxKeepDim});
  ASSERT_EQ(t1c2.Size(0), 5);
  ASSERT_EQ(t1c2.Size(1), 6);
  ASSERT_EQ(t1c2.Size(2), t1.Size(0));
  for (index_t i = 0; i < t1c2.Size(0); i++) {
    for (index_t j = 0; j < t1c2.Size(1); j++) {
      for (index_t k = 0; k < t1c2.Size(2); k++) {
        ASSERT_EQ(t1c2(i, j, k), t1(k));
      }
    }
  }

  auto t1c3 = t1.Clone<4>({5, 6, 7, matxKeepDim});
  ASSERT_EQ(t1c3.Size(0), 5);
  ASSERT_EQ(t1c3.Size(1), 6);
  ASSERT_EQ(t1c3.Size(2), 7);
  ASSERT_EQ(t1c3.Size(3), t1.Size(0));
  for (index_t i = 0; i < t1c3.Size(0); i++) {
    for (index_t j = 0; j < t1c3.Size(1); j++) {
      for (index_t k = 0; k < t1c3.Size(2); k++) {
        for (index_t l = 0; l < t1c3.Size(3); l++) {
          ASSERT_EQ(t1c3(i, j, k, l), t1(l));
        }
      }
    }
  }

  // clone t2 across 0/1 dim
  auto t2c1 = t2.Clone<3>({5, matxKeepDim, matxKeepDim});
  ASSERT_EQ(t2c1.Size(0), 5);
  for (index_t i = 0; i < t2c1.Size(0); i++) {
    for (index_t j = 0; j < t2c1.Size(1); j++) {
      for (index_t k = 0; k < t2c1.Size(2); k++) {
        ASSERT_EQ(t2c1(i, j, k), t2(j, k));
      }
    }
  }

  auto t2c2 = t2.Clone<4>({5, 6, matxKeepDim, matxKeepDim});
  ASSERT_EQ(t2c2.Size(0), 5);
  ASSERT_EQ(t2c2.Size(1), 6);
  for (index_t i = 0; i < t2c2.Size(0); i++) {
    for (index_t j = 0; j < t2c2.Size(1); j++) {
      for (index_t k = 0; k < t2c2.Size(2); k++) {
        for (index_t l = 0; l < t2c2.Size(3); l++) {
          ASSERT_EQ(t2c2(i, j, k, l), t2(k, l));
        }
      }
    }
  }

  // clone t3 across 0 dim
  auto t3c1 = t3.Clone<4>({5, matxKeepDim, matxKeepDim, matxKeepDim});
  ASSERT_EQ(t3c1.Size(0), 5);
  for (index_t i = 0; i < t3c1.Size(0); i++) {
    for (index_t j = 0; j < t3c1.Size(1); j++) {
      for (index_t k = 0; k < t3c1.Size(2); k++) {
        for (index_t l = 0; l < t3c1.Size(3); l++) {
          ASSERT_EQ(t3c1(i, j, k, l), t3(j, k, l));
        }
      }
    }
  }

  MATX_EXIT_HANDLER();
}

TEST(ViewTests, Random)
{
  MATX_ENTER_HANDLER();
  {
    index_t count = 100;
    randomGenerator_t<float> rfloat(count * count * count, 0);
    auto t3fu = rfloat.GetTensorView<3>({count, count, count}, UNIFORM);
    auto t3fn = rfloat.GetTensorView<3>({count, count, count}, NORMAL);

    tensor_t<float, 3> t3f({count, count, count});

    (t3f = -1000000).run();
    (t3f = t3fu).run();
    t3f.PrefetchHost(0);
    hipDeviceSynchronize();

    float total = 0;
    for (index_t i = 0; i < count; i++) {
      for (index_t j = 0; j < count; j++) {
        for (index_t k = 0; k < count; k++) {
          float val = t3f(i, j, k) - 0.5f; // mean centered at zero
          ASSERT_NE(val, -1000000);
          total += val;
          ASSERT_LE(val, 0.5f);
          ASSERT_LE(-0.5f, val);
        }
      }
    }

    ASSERT_LT(fabs(total / (count * count * count)), .05);

    (t3f = -1000000).run();
    (t3f = t3fn).run();
    t3f.PrefetchHost(0);
    hipDeviceSynchronize();

    total = 0;

    for (index_t i = 0; i < count; i++) {
      for (index_t j = 0; j < count; j++) {
        for (index_t k = 0; k < count; k++) {
          float val = t3f(i, j, k);
          ASSERT_NE(val, -1000000);
          total += val;
        }
      }
    }

    ASSERT_LT(fabs(total / (count * count * count)), .15);
  }
  MATX_EXIT_HANDLER();
}


TYPED_TEST(ViewTestsComplex, RealComplexView)
{
  MATX_ENTER_HANDLER();
  tensor_t<TypeParam, 1> tc({10});
  auto tr = tc.RealView();
  auto ti = tc.ImagView();

  for (int i = 0; i < 10; i++) {
    TypeParam val(
        static_cast<promote_half_t<typename TypeParam::value_type>>(i),
        static_cast<promote_half_t<typename TypeParam::value_type>>(i + 10));
    tc(i) = val;
  }

  for (int i = 0; i < 10; i++) {
    ASSERT_EQ((float)tc(i).real(), (float)tr(i));
    ASSERT_EQ((float)tc(i).imag(), (float)ti(i));
  }
  MATX_EXIT_HANDLER();
}
