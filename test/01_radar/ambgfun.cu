#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "utilities.h"
#include "gtest/gtest.h"

using namespace matx;
using complex = cuda::std::complex<float>;

class RadarAmbiguityFunction : public ::testing::Test {
protected:
  void SetUp() override
  {

    pb = std::make_unique<detail::MatXPybind>();
    pb->InitAndRunTVGenerator<complex>("01_radar", "ambgfun", "run",
                                       {sig_size});

    pb->NumpyToTensorView(xv, "x");
  }

  void TearDown() { pb.reset(); }

  index_t sig_size = 16;
  tensor_t<complex, 1> xv{{sig_size}};
  std::unique_ptr<detail::MatXPybind> pb;
};

TEST_F(RadarAmbiguityFunction, Cut2D)
{
  MATX_ENTER_HANDLER();

  tensor_t<float, 2> amf2dv(
      {2 * sig_size - 1,
       (index_t)pow(2, std::ceil(std::log2(2 * sig_size - 1)))});

  ambgfun(amf2dv, xv, 1e3, AMGBFUN_CUT_TYPE_2D, 1.0);
  MATX_TEST_ASSERT_COMPARE(pb, amf2dv, "amf_2d", 0.01);

  MATX_EXIT_HANDLER();
}

TEST_F(RadarAmbiguityFunction, CutDelay)
{
  MATX_ENTER_HANDLER();

  tensor_t<float, 2> amf_delay_v(
      {1, (index_t)pow(2, std::ceil(std::log2(2 * sig_size - 1)))});

  ambgfun(amf_delay_v, xv, 1e3, AMGBFUN_CUT_TYPE_DELAY, 1.0);

  auto delay1d = amf_delay_v.Slice<1>({0, 0}, {matxDropDim, matxEnd});
  MATX_TEST_ASSERT_COMPARE(pb, delay1d, "amf_delay", 0.01);

  MATX_EXIT_HANDLER();
}

TEST_F(RadarAmbiguityFunction, CutDoppler)
{
  MATX_ENTER_HANDLER();

  tensor_t<float, 2> amf_doppler_v({1, xv.Size(0) * 2 - 1});

  ambgfun(amf_doppler_v, xv, 1e3, AMGBFUN_CUT_TYPE_DOPPLER,
                  1.0);

  auto doppler1d = amf_doppler_v.Slice<1>({0, 0}, {matxDropDim, matxEnd});
  MATX_TEST_ASSERT_COMPARE(pb, doppler1d, "amf_doppler", 0.01);

  MATX_EXIT_HANDLER();
}
