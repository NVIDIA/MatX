#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "test_types.h"
#include "utilities.h"
#include "gtest/gtest.h"

using namespace matx;
constexpr index_t m = 100;
constexpr index_t n = 50;

template <typename T> class SVDSolverTest : public ::testing::Test {
protected:
  void SetUp() override
  {
    pb = std::make_unique<detail::MatXPybind>();
    pb->InitAndRunTVGenerator<T>("00_solver", "svd", "run", {m, n});
    pb->NumpyToTensorView(Av, "A");
  }

  void TearDown() { pb.reset(); }

  std::unique_ptr<detail::MatXPybind> pb;
  tensor_t<T, 2> Av{{m, n}};
  tensor_t<T, 2> Atv{{n, m}};
  tensor_t<T, 1> Sv{{std::min(m, n)}};
  tensor_t<T, 2> Uv{{m, m}};
  tensor_t<T, 2> Vv{{n, n}};

  tensor_t<T, 2> Sav{{m, n}};
  tensor_t<T, 2> Uav{{m, m}};
  tensor_t<T, 2> Vav{{n, n}};

  // Used only for validation
  tensor_t<T, 2> tmpV{{m, n}};
};

template <typename TensorType>
class SVDSolverTestNonComplexFloatTypes : public SVDSolverTest<TensorType> {
};

TYPED_TEST_SUITE(SVDSolverTestNonComplexFloatTypes,
                 MatXFloatNonComplexNonHalfTypes);

TYPED_TEST(SVDSolverTestNonComplexFloatTypes, SVDBasic)
{
  MATX_ENTER_HANDLER();

  // cuSolver only supports col-major solving today, so we need to transpose,
  // solve, then transpose again to compare to Python
  transpose(this->Atv, this->Av, 0);

  auto Atv2 = this->Atv.View({m, n});
  svd(this->Uv, this->Sv, this->Vv, Atv2);

  hipStreamSynchronize(0);

  // Since SVD produces a solution that's not necessarily unique, we cannot
  // compare against Python output. Instead, we just make sure that A = U*S*V'.
  // However, U and V are in column-major format, so we have to transpose them
  // back to verify the identity.
  transpose(this->Uav, this->Uv, 0);
  transpose(this->Vav, this->Vv, 0);

  // Zero out s
  (this->Sav = zeros({m, n})).run();
  hipStreamSynchronize(0);

  // Construct S matrix since it's just a vector from cuSolver
  for (index_t i = 0; i < n; i++) {
    this->Sav(i, i) = this->Sv(i);
  }

  hipStreamSynchronize(0);

  matmul(this->tmpV, this->Uav, this->Sav); // U * S
  matmul(this->Sav, this->tmpV, this->Vav); // (U * S) * V'
  hipStreamSynchronize(0);

  for (index_t i = 0; i < this->Av.Size(0); i++) {
    for (index_t j = 0; j < this->Av.Size(1); j++) {
      ASSERT_NEAR(this->Av(i, j), this->Sav(i, j), 0.001) << i << " " << j;
    }
  }

  MATX_EXIT_HANDLER();
}
