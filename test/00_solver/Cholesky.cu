////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "test_types.h"
#include "utilities.h"
#include "gtest/gtest.h"

using namespace matx;
constexpr int dim_size = 100;

template <typename T> class CholSolverTest : public ::testing::Test {
protected:
  void SetUp() override
  {
    pb = std::make_unique<detail::MatXPybind>();
    pb->InitAndRunTVGenerator<T>("00_solver", "cholesky", "run", {dim_size});
    pb->NumpyToTensorView(Bv, "B");
    pb->NumpyToTensorView(Lv, "L");
  }

  void TearDown() { pb.reset(); }

  std::unique_ptr<detail::MatXPybind> pb;
  tensor_t<T, 2> Bv{{dim_size, dim_size}};
  tensor_t<T, 2> Lv{{dim_size, dim_size}};
};

template <typename TensorType>
class CholSolverTestNonComplexFloatTypes : public CholSolverTest<TensorType> {
};

TYPED_TEST_SUITE(CholSolverTestNonComplexFloatTypes,
                 MatXFloatNonComplexNonHalfTypes);

TYPED_TEST(CholSolverTestNonComplexFloatTypes, CholeskyBasic)
{
  MATX_ENTER_HANDLER();

  chol(this->Bv, this->Bv, 0, HIPBLAS_FILL_MODE_LOWER);
  hipStreamSynchronize(0);

  // Cholesky only saves the upper triangle by default, and the lower triangle
  // is garbage. Python saves the opposite.
  for (index_t i = 0; i < this->Bv.Size(0); i++) {
    for (index_t j = 0; j <= i; j++) {
      ASSERT_NEAR(this->Bv(i, j), this->Lv(i, j), 0.001);
    }
  }

  MATX_EXIT_HANDLER();
}
