////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "test_types.h"
#include "utilities.h"
#include "gtest/gtest.h"

using namespace matx;
constexpr int m = 15;

template <typename T> class DetSolverTest : public ::testing::Test {
protected:
  void SetUp() override
  {
    pb = std::make_unique<detail::MatXPybind>();
    pb->InitAndRunTVGenerator<T>("00_solver", "det", "run", {m});
    pb->NumpyToTensorView(Av, "A");
  }

  void TearDown() { pb.reset(); }

  std::unique_ptr<detail::MatXPybind> pb;
  tensor_t<T, 2> Av{{m, m}};
  tensor_t<T, 2> Atv{{m, m}};
  tensor_t<T, 0> detv{};
};

template <typename TensorType>
class DetSolverTestNonComplexFloatTypes : public DetSolverTest<TensorType> {
};

TYPED_TEST_SUITE(DetSolverTestNonComplexFloatTypes,
                 MatXFloatNonComplexNonHalfTypes);

TYPED_TEST(DetSolverTestNonComplexFloatTypes, Determinant)
{
  MATX_ENTER_HANDLER();

  // cuSolver only supports col-major solving today, so we need to transpose,
  // solve, then transpose again to compare to Python
  transpose(this->Atv, this->Av, 0);

  det(this->detv, this->Atv);
  transpose(this->Av, this->Atv, 0); // Transpose back to row-major
  hipStreamSynchronize(0);

  MATX_TEST_ASSERT_COMPARE(this->pb, this->detv, "det", 0.1);

  MATX_EXIT_HANDLER();
}
