#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "assert.h"
#include "matx.h"
#include "test_types.h"
#include "utilities.h"
#include "gtest/gtest.h"

using namespace matx;
constexpr int m = 100;
constexpr int n = 50;

template <typename T> class QRSolverTest : public ::testing::Test {
protected:
  using dtype = float;
  using GTestType = std::tuple_element_t<0, T>;
  using GExecType = std::tuple_element_t<1, T>;   
   
  void SetUp() override
  {
    pb = std::make_unique<detail::MatXPybind>();
    pb->InitAndRunTVGenerator<GTestType>("00_solver", "qr", "run", {m, n});
    pb->NumpyToTensorView(Av, "A");
    pb->NumpyToTensorView(Qv, "Q");
    pb->NumpyToTensorView(Rv, "R");
  }

  void TearDown() { pb.reset(); }

  std::unique_ptr<detail::MatXPybind> pb;
  GExecType exec{};   
  tensor_t<GTestType, 2> Av{{m, n}};
  tensor_t<GTestType, 2> Atv{{n, m}};
  tensor_t<GTestType, 1> TauV{{std::min(m, n)}};
  tensor_t<GTestType, 2> Qv{{m, std::min(m, n)}};
  tensor_t<GTestType, 2> Rv{{std::min(m, n), n}};
};

template <typename TensorType>
class QRSolverTestNonComplexFloatTypes : public QRSolverTest<TensorType> {
};

TYPED_TEST_SUITE(QRSolverTestNonComplexFloatTypes,
                 MatXFloatNonComplexNonHalfTypesCUDAExec);

TYPED_TEST(QRSolverTestNonComplexFloatTypes, QRBasic)
{
  MATX_ENTER_HANDLER();

  // example-begin cusolver_qr-test-1
  // cuSolver only supports col-major solving today, so we need to transpose,
  // solve, then transpose again to compare to Python
  (mtie(this->Av, this->TauV) = cusolver_qr(this->Av)).run(this->exec);
  // example-end cusolver_qr-test-1
  hipStreamSynchronize(0);

  // For now we're only verifying R. Q is a bit more complex to compute since
  // cuSolver/BLAS don't return Q, and instead return Householder reflections
  // that are used to compute Q. Eventually compute Q here and verify
  for (index_t i = 0; i < this->Av.Size(0); i++) {
    for (index_t j = 0; j < this->Av.Size(1); j++) {
      // R is stored only in the top triangle of A
      if (i <= j) {
        ASSERT_NEAR(this->Av(i, j), this->Rv(i, j), 0.001);
      }
    }
  }

  MATX_EXIT_HANDLER();
}
