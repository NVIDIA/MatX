////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include <matx.h>


// BUILD NOTES: TO build, include the path to the eigen in cmake with the variable eigen_DIR="Path/To/Eigen"
#ifdef USE_EIGEN
  #include <Eigen/Dense>
#endif

#include <iostream>



int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  int dimX = 3;
  int dimY = 3;

  matx::cudaExecutor exec{};

  ///////////////////////////////////////////////////////////////////////////////
  //////////////               Eigen Test Data Setup               //////////////
  ///////////////////////////////////////////////////////////////////////////////
#ifdef USE_EIGEN
  
  typedef Eigen::Matrix<double, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> MatrixXdRowMajor; // define a custom type that is aligned to MatX row-Major.
  
  Eigen::MatrixXd  a(dimX, dimY);
  MatrixXdRowMajor b(dimX, dimY);
  Eigen::RowVectorXd rowVec(dimX);
  Eigen::Matrix<std::complex<double>, 2, 2> complexMatrix;
  Eigen::MatrixXf matrix10x10(10, 10);
#endif
  ///////////////////////////////////////////////////////////////////////////////
  //////////////                MatX Test Data Setup               //////////////
  ///////////////////////////////////////////////////////////////////////////////
  auto aTensor  = matx::make_tensor<double>({dimX,dimY});
  auto bTensor  = matx::make_tensor<double>({dimX,dimY});
  auto tensor1D = matx::make_tensor<double>({dimX});
  auto complexTensor = matx::make_tensor<cuda::std::complex<double>>({2,2});
  auto matTensor10x10 = matx::make_tensor<float>({10,10});


  ///////////////////////////////////////////////////////////////////////////////
  //////////////                   Initialize Data                 //////////////
  ///////////////////////////////////////////////////////////////////////////////
#ifdef USE_EIGEN
  std::cout <<"!!!!!!!!! Using Eigen in Test !!!!!!!!!" << std ::endl;
  // Initialize with random values
  a.setRandom();
  b.setRandom();
  matrix10x10.setRandom();

  rowVec << 1, 2, 3;

  complexMatrix(0, 0) = std::complex<double>(1.0, 2.0);
  complexMatrix(0, 1) = std::complex<double>(2.0, 3.0);
  complexMatrix(1, 0) = std::complex<double>(3.0, 4.0);
  complexMatrix(1, 1) = std::complex<double>(4.0, 5.0);
  
#else
  std::cout <<"!!!!!!!!! Eigen NOT USED in Test !!!!!!!!!" << std ::endl;
  // provide data in tensors if eigen is not used
  (aTensor = matx::random<double>({dimX, dimY}, matx::UNIFORM)).run(exec);
  (bTensor = matx::random<double>({dimX, dimY}, matx::UNIFORM)).run(exec);
  (complexTensor = matx::random<cuda::std::complex<double>>({2, 2}, matx::UNIFORM)).run(exec);
  (matTensor10x10 = matx::random<double>({10, 10}, matx::UNIFORM)).run(exec);

#endif




  ///////////////////////////////////////////////////////////////////////////////
  //////////////              Copy Eigen inputs to MatX            //////////////
  ///////////////////////////////////////////////////////////////////////////////
#ifdef USE_EIGEN
  hipMemcpy(aTensor.Data(), a.data(), sizeof(double) * dimX * dimY, hipMemcpyHostToDevice);
  hipMemcpy(bTensor.Data(), b.data(), sizeof(double) * dimX * dimY, hipMemcpyHostToDevice);
  hipMemcpy(complexTensor.Data(), complexMatrix.data(), sizeof(std::complex<double>)*2*2, hipMemcpyHostToDevice);
  hipMemcpy(matTensor10x10.Data(), matrix10x10.data(), sizeof(float)*10*10, hipMemcpyHostToDevice);

  (aTensor = matx::transpose(aTensor)).run(exec);
  // (bTensor = matx::transpose(bTensor)).run(exec); // do not need to transpose because b has the same layout
  (complexTensor = matx::transpose(complexTensor)).run(exec);
  (matTensor10x10 = matx::transpose(matTensor10x10)).run(exec);
#endif

  tensor1D(0) = 1;
  tensor1D(1) = 2;
  tensor1D(2) = 3;
  exec.sync();

  // slower alternative of copying per-element
  // for(int curX=0; curX<dimX; curX++)
  // {
  //   for(int curY=0; curY<dimY; curY++)
  //   {
  //     aTensor(curX,curY) = a(curX,curY);
  //     bTensor(curX,curY) = b(curX,curY);
  //   }
  // }


  ///////////////////////////////////////////////////////////////////////////////
  //////////////                 Operation Examples                //////////////
  ///////////////////////////////////////////////////////////////////////////////


  //
  // Data Mapping Example
  // 
#ifdef USE_EIGEN  
  std::cout << "=================== Data Map Example ===================" << std::endl;
  double *raw_data;
  // memory could be any type of allocation, but choosing to use managed memory so it's valid on the host and device (this does cost performance)
  hipMallocManaged((void**)&raw_data, dimX*dimY * sizeof(double)); 
  
  for(int i=0; i < dimX * dimY; i++)
  {
    raw_data[i] = 0.1 + i * 0.1;
  }
  
  // map user memory into Eigen Matrix
  Eigen::Map<MatrixXdRowMajor> mappedMatrix(raw_data, dimX, dimY);
  std::cout << "Eigen Mapped Data :\n" << mappedMatrix << std::endl;

  // map user memory into Eigen Matrix
  auto mappedTensor = matx::make_tensor(raw_data, {dimX, dimY}, false); // create MatX tensor with non-owning user allocated memory
  matx::print(mappedTensor);
  
  // modify the data from each of the references  
  raw_data[4] = 117;
  mappedMatrix(0,1) = 42;
  mappedTensor(2,1) = 87;
  
  // print modified data
  std::cout << "Eigen Mapped Data After Modified :\n" << mappedMatrix << std::endl;
  matx::print(mappedTensor);
#endif 

  //
  // Basic Indexing
  //
  std::cout << "=================== Indexing ===================" << std::endl;
#ifdef USE_EIGEN
  std::cout << "eigen a(1,2) = " <<  a(1,2) << std::endl; 
#endif  

  std::cout << "MatX  a(1,2) = " << aTensor(1,2) << std::endl;


  //
  // Add A and B
  //
  std::cout << "=================== Addition ===================" << std::endl;
#ifdef USE_EIGEN
  Eigen::MatrixXd addResult = a + b; 
  std::cout << "A + B = \n" << addResult << std::endl; 
#endif  

  auto addTensor = aTensor + bTensor;
  matx::print(addTensor);


  //
  // Element-Wise Multiply A and B
  //
  std::cout << "=================== Element-Wise Multiply ===================" << std::endl;
#ifdef USE_EIGEN
  Eigen::MatrixXd elementWise = a.cwiseProduct(b); 
  std::cout << "A .* B = \n" << elementWise << std::endl; 
#endif  

  auto elementWiseTensor = aTensor*bTensor;
  matx::print(elementWiseTensor);


  //
  // Divide A and B
  //
  std::cout << "=================== Element-Wise Division ===================" << std::endl;
#ifdef USE_EIGEN
  Eigen::MatrixXd divResult = a.cwiseQuotient(b); 
  std::cout << "A / B = \n" << divResult << std::endl; 
#endif  

  auto divResultTensor = aTensor / bTensor;
  matx::print(divResultTensor);


  //
  // Slice (Continuous)
  //
  std::cout << "=================== Continuous Slice ===================" << std::endl;
#ifdef USE_EIGEN
  Eigen::Matrix2d aSlice = a.block(0, 0, 2, 2);       
  std::cout << "A Sliced: \n" << aSlice << std::endl; 
#endif  

  auto aSliceTensor = matx::slice<2>(aTensor,{0,0},{2,2});
  matx::print(aSliceTensor);


  //
  // Slice (Strided)
  //
  std::cout << "=================== Strided Slice ===================" << std::endl;
#ifdef USE_EIGEN
  std::cout << "Original matrix10x10:\n" << matrix10x10 << "\n\n"; 
  // Define the starting point, number of elements to select, and strides for both rows and columns
  // int startRow = 0, startCol = 0; // Starting index for rows and columns
  // int rowStride = 3, colStride = 2; // Stride along rows and columns
  // int numRows = 5; // Calculate the number of rows, considering every second element
  // int numCols = 3; // Grab every third item until the 8th item (0, 3, 6)

  // Create a Map with Stride to access the elements
  Eigen::Map<Eigen::MatrixXf, 0, Eigen::Stride<Eigen::Dynamic, Eigen::Dynamic>>                        
  strided(matrix10x10.data() + 0 * matrix10x10.outerStride() + 0,                                                
          5, 3,                                                                                        
          Eigen::Stride<Eigen::Dynamic, Eigen::Dynamic>(3 * matrix10x10.outerStride(), 2));                 

  // Print the strided matrix10x10
  std::cout << "Strided matrix10x10:\n" << strided << "\n";  
#endif  

  auto slicedMat = matx::slice(matTensor10x10, {0,0}, {matx::matxEnd,9}, {2,3});
  matx::print(slicedMat);


  //
  // Clone
  //
  std::cout << "=================== Clone ===================" << std::endl;
#ifdef USE_EIGEN
  // Use the replicate function to create a 5x5 matrix by replicating the 1x5 matrix
  Eigen::MatrixXd mat = rowVec.replicate(3, 1);         
  std::cout << "1D Cloned to 2D \n" << mat << std::endl;
#endif  

  auto cloned3Tensor = matx::clone<2>(tensor1D, {3, matx::matxKeepDim});
  matx::print(cloned3Tensor);


  //
  // Slice Row
  //
  std::cout << "=================== Slice Row ===================" << std::endl;
#ifdef USE_EIGEN
  Eigen::RowVector3d row = a.row(1);               
  std::cout << "Sliced Row \n" << row << std::endl;
#endif  

  auto rowSlice = matx::slice<1>(aTensor, {1, 0}, {matx::matxDropDim, matx::matxEnd});
  matx::print(rowSlice);


  //
  // Permute Rows
  //
  std::cout << "=================== Permute Rows ===================" << std::endl;
#ifdef USE_EIGEN
  std::cout << "Original Matrix:\n" << a << std::endl;  
  // Define a permutation a
  Eigen::PermutationMatrix<3> perm;    
  perm.indices() << 2, 1, 0;  // This permutation swaps the first and third rows
  // Apply the permutation to the rows
  Eigen::Matrix3d permutedMatrix = perm * a;
  std::cout << "Permuted Matrix (Rows):\n" << permutedMatrix << std::endl;
#endif  

  // Define a permutation a
  auto permVec = matx::make_tensor<int>({dimX});
  permVec(0) = 2;
  permVec(1) = 1;
  permVec(2) = 0;
  // Apply the permutation to the rows
  auto permTensor = matx::remap<0>(aTensor, permVec);
  matx::print(permTensor);


  //
  // Permutation Dimensions
  //
  std::cout << "=================== Permute Dimension ===================" << std::endl;
  // Unsupported by eigen
  auto permA = permute(aTensor, {1,0});
  matx::print(permA);

  //
  // Get Real Value
  //
  std::cout << "=================== Get Real Values ===================" << std::endl;
#ifdef USE_EIGEN
  std::cout << "Original Complex Matrix:\n" << complexMatrix << std::endl; 

  // Extract and output the real part of the complex matrix
  Eigen::Matrix<double, 2, 2> realMatrix = complexMatrix.real();    
  std::cout << "Real Part of Matrix:\n" << realMatrix << std::endl; 
#endif  

  auto realTensor = matx::real(complexTensor);
  matx::print(realTensor);


  //
  // Multiply A and B
  //
  std::cout << "=================== Matrix Multiply ===================" << std::endl;
#ifdef USE_EIGEN
  Eigen::MatrixXd multResult = a * b;                    
  std::cout << "A * B = \n" << multResult << std::endl;  
#endif  

  auto multResultTensor=matmul(aTensor,bTensor);
  matx::print(multResultTensor);


  //
  // inverse Matrix
  //
  std::cout << "=================== Invert Matrix ===================" << std::endl;
#ifdef USE_EIGEN
  // Eigen::MatrixXd inverseMatrix = a.inverse();                                              // current bug where .run(exec) in inverse is ambiguous, so cannot be used with MatX
  // std::cout << "Inverse of the Real Part:\n" << inverseMatrix << std::endl; // current bug where .run(exec) in inverse is ambiguous, so cannot be used with MatX
#endif  

  auto invTensor = matx::inv(aTensor);
  matx::print(invTensor);

  //
  // 1D FFT
  //
  // Unsupported by eigen

  //
  // 1D IFFT
  //
  // Unsupported by eigen

  //
  // 2D FFT
  //
  // Unsupported by eigen

  //
  // 2D IFFT
  //
  // Unsupported by eigen

  //
  // Covariance
  //
  // Unsupported by eigen

  return 0;
}
