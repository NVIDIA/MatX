////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2023, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "matx.h"
#include <cassert>
#include <cstdio>
#include <cuda/std/ccomplex>

using namespace matx;

/**
 * PWelch Example
 *
 * This example shows how to estimate the power spectral density of a signal using the pwelch() operator
 * using Welch's method.
 *
 */

int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();
  using complex = cuda::std::complex<float>;

  float exec_time_ms;
  const int num_iterations = 100;
  index_t signal_size = 256;
  index_t nperseg = 32;
  index_t nfft = nperseg;
  index_t noverlap = 8;
  float ftone = 3.0;
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  cudaExecutor exec{stream};

  // Create input signal as a complex exponential
  auto sample_index_range = range<0>({signal_size},0.f,1.f);
  auto phase = 2.f * static_cast<float>(M_PI) * ftone * sample_index_range / static_cast<float>(nfft);
  auto tmp_x = expj(phase);
  auto x = make_tensor<complex>({signal_size});
  (x = tmp_x).run(exec); // pre-compute x, tmp_x is otherwise lazily evaluated

  // Create output tensor
  auto Pxx  = make_tensor<typename complex::value_type>({nfft});

  // Run one time to pre-cache the FFT plan
  (Pxx = pwelch(x, nperseg, noverlap, nfft)).run(exec);
  exec.sync();

  // Start the timing
  hipEventRecord(start, stream);

  // Start the timing
  hipEventRecord(start, stream);

  for (int iteration = 0; iteration < num_iterations; iteration++) {
    // Use the PWelch operator
    (Pxx = pwelch(x, nperseg, noverlap, nfft)).run(exec);
  }

  hipEventRecord(stop, stream);
  exec.sync();
  hipEventElapsedTime(&exec_time_ms, start, stop);

  printf("Output Pxx:\n");
  print(Pxx);
  printf("PWelchOp avg runtime = %.3f ms\n", exec_time_ms / num_iterations);

  CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
  return 0;
}
