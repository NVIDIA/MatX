////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2023, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "matx.h"
#include <cassert>
#include <cstdio>
#include <cuda/std/ccomplex>

using namespace matx;

/**
 * PWelch Example
 *
 * This example shows how to estimate the power spectral density of a signal using the pwelch() operator
 * using Welch's method.
 *
 */

int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();
  using complex = cuda::std::complex<float>;

  const int num_iterations = 500;
  index_t signal_size = 256000;
  index_t nperseg = 512;
  index_t noverlap = 256;
  index_t nfft = 65536;

  float ftone = 2048.0;
  hipStream_t stream;
  hipStreamCreate(&stream);
  cudaExecutor exec{stream};

  // Create input signal as a complex exponential
  auto sample_index_range = range<0>({signal_size},0.f,1.f);
  auto phase = 2.f * static_cast<float>(M_PI) * ftone * sample_index_range / static_cast<float>(nfft);
  auto tmp_x = expj(phase);
  auto x = make_tensor<complex>({signal_size});
  (x = tmp_x).run(exec); // pre-compute x, tmp_x is otherwise lazily evaluated

  // Create window
  auto w = make_tensor<float>({nperseg});
  (w = flattop<0>({nperseg})).run(exec);

  // Create output tensor
  auto Pxx  = make_tensor<typename complex::value_type>({nfft});

  // Run one time to pre-cache the FFT plan
  (Pxx = pwelch(x, w, nperseg, noverlap, nfft)).run(exec);
  exec.sync();

  // Start the timing
  exec.start_timer();

  for (int iteration = 0; iteration < num_iterations; iteration++) {
    // Use the PWelch operator
    (Pxx = pwelch(x, w, nperseg, noverlap, nfft)).run(exec);
  }
  exec.sync();
  exec.stop_timer();

  printf("Pxx(0) = %f\n", Pxx(0));
  printf("Pxx(ftone) = %f\n", Pxx(2048));
  printf("PWelchOp avg runtime = %.3f ms\n", exec.get_time_ms() / num_iterations);

  MATX_CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
  return 0;
}
