////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "matx.h"
#include "mvdr_beamformer.h"
#include <cassert>
#include <cstdio>
#include <memory>
#include <stdlib.h>

using namespace matx;

int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();

  uint32_t num_beams = 60;
  uint32_t num_el = 6;
  uint32_t data_len = 65536;
  uint32_t snap_len = 2 * num_el;

  constexpr uint32_t num_iterations = 1;
  float time_ms;

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  auto mvdr = MVDRBeamformer(num_beams, num_el, data_len, snap_len);

  auto v = mvdr.GetV();
  for (index_t i = 0; i < v.Size(0); i++) {
    for (index_t j = 0; j < v.Size(1); j++) {
      v(i, j) = {static_cast<float>(rand()) / static_cast<float>(RAND_MAX),
                 static_cast<float>(rand()) / static_cast<float>(RAND_MAX)};
    }
  }

  auto invec = mvdr.GetInVec();
  for (index_t i = 0; i < invec.Size(0); i++) {
    for (index_t j = 0; j < invec.Size(1); j++) {
      invec(i, j) = {static_cast<float>(rand()) / static_cast<float>(RAND_MAX),
                     static_cast<float>(rand()) / static_cast<float>(RAND_MAX)};
    }
  }

  mvdr.Prefetch(stream);

  hipEventRecord(start, stream);

  for (uint32_t i = 0; i < num_iterations; i++) {
    mvdr.Run(stream);
  }

  hipEventRecord(stop, stream);
  hipStreamSynchronize(stream);
  hipEventElapsedTime(&time_ms, start, stop);

  printf("MVDR Kernel Time = %.2fms per iteration\n", time_ms / num_iterations);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);
  CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
}