#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include <cassert>
#include <cstdio>
#include <math.h>
#include <memory>

#include "matx.h"

using namespace matx;
#define FFT_TYPE HIPFFT_C2C

/** Create a spectrogram of a signal
 *
 * This example creates a set of data representing signal power versus frequency
 * and time. Traditionally the signal power is plotted as the Z dimension using
 * color, and time/frequency are the X/Y axes. The time taken to run the
 * spectrogram is computed, and a simple scatter plot is output. This version
 * does not use CUDA graphs, and kernel launches are launched in a loop
 * asynchronously from the host. See spectrogram_graph.cu for a version using
 * CUDA graphs, which gives a performance boost by launching a graph once per
 * iteration instead of separate kernels.
 */

int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();

  using complex = cuda::std::complex<float>;

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float fs = 10000;
  constexpr index_t N = 100000;
  float amp = static_cast<float>(2 * sqrt(2));
  constexpr index_t nperseg = 256;
  constexpr index_t nfft = 256;
  constexpr index_t noverlap = nperseg / 8;
  constexpr index_t nstep = nperseg - noverlap;
  constexpr uint32_t num_iterations = 100;
  float time_ms;

  std::array<index_t, 1> num_samps{N};
  std::array<index_t, 1> half_win{nfft / 2 + 1};
  std::array<index_t, 1> s_time_shape{(N - noverlap) / nstep};

  auto time = make_tensor<float>({N});
  auto modulation = make_tensor<float>({N});
  auto carrier = make_tensor<float>({N});
  auto noise = make_tensor<float>({N});
  auto x = make_tensor<float>({N});

  auto freqs = make_tensor<float>({nfft / 2 + 1});
  auto fftStackedMatrix = make_tensor<complex>({(N - noverlap) / nstep, nfft / 2 + 1});
  auto s_time = make_tensor<float>({(N - noverlap) / nstep});

  randomGenerator_t<float> randData({N}, 0);
  auto randDataView = randData.GetTensorView<1>(num_samps, NORMAL);

  // Set up all static buffers
  // time = np.arange(N) / float(fs)
  (time = linspace<0>(num_samps, 0.0f, static_cast<float>(N) - 1.0f) / fs)
      .run(stream);
  // mod = 500 * np.cos(2*np.pi*0.25*time)
  (modulation = 500 * cos(2 * M_PI * 0.25 * time)).run(stream);
  // carrier = amp * np.sin(2*np.pi*3e3*time + modulation)
  (carrier = amp * sin(2 * M_PI * 3000 * time + modulation)).run(stream);
  // noise = 0.01 * fs / 2 * np.random.randn(time.shape)
  (noise = sqrt(0.01 * fs / 2) * randDataView).run(stream);
  // noise *= np.exp(-time/5)
  (noise = noise * exp(-1.0f * time / 5.0f)).run(stream);
  // x = carrier + noise
  (x = carrier + noise).run(stream);

  for (uint32_t i = 0; i < num_iterations; i++) {
    if (i == 2) { // Start timer on third loop to allow generation of plot
      hipEventRecord(start, stream);
    }

    // DFT Sample Frequencies (rfftfreq)
    (freqs = (1.0 / (static_cast<float>(nfft) * 1 / fs)) *
               linspace<0>(half_win, 0.0f, static_cast<float>(nfft) / 2.0f))
        .run(stream);

    // Create overlapping matrix of segments.
    auto stackedMatrix = x.OverlapView({nperseg}, {nstep});
    // FFT along rows
    fft(fftStackedMatrix, stackedMatrix, 0, stream);
    // Absolute value
    (fftStackedMatrix = conj(fftStackedMatrix) * fftStackedMatrix)
        .run(stream);
    // Get real part and transpose
    auto Sxx = fftStackedMatrix.RealView().Permute({1, 0});

    // Spectral time axis
    (s_time = linspace<0>(s_time_shape, static_cast<float>(nperseg) / 2.0f,
                           static_cast<float>(N - nperseg) / 2.0f + 1) /
                fs)
        .run(stream);

    if (i == 1) {
#if MATX_ENABLE_VIZ
      // Generate a spectrogram visualization using a contour plot
      viz::contour(time, freqs, Sxx);
#else
      printf("Not outputting plot since visualizations disabled\n");
#endif    
    }
  }

  hipEventRecord(stop, stream);
  hipStreamSynchronize(stream);
  hipEventElapsedTime(&time_ms, start, stop);

  printf("Spectrogram Time Without Graphs = %.2fus per iteration\n",
         time_ms * 1e3 / num_iterations);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);
  CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
}
