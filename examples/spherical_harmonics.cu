#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "matx.h"
#include <cassert>
#include <cstdio>
#include <math.h>

using namespace matx;

template<typename T> T factorial(int N) {
  T prod = 1;
  for(int i=2; i<=N; i++) {
    prod = prod * i;
  }
  return prod;
}

int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();

  cudaExecutor exec{};

  using ValueType = double;

  int l = 3;
  int m = 2;
  int n = 60;
  ValueType dx = M_PI/n;

  auto col = range<0>({n+1},ValueType(0), ValueType(dx));
  auto az = range<0>({2*n+1}, ValueType(0), ValueType(dx));

  auto [phi, theta] = meshgrid(az, col);

  auto Plm = lcollapse<3>(legendre(l, m, cos(theta)));
 
  ValueType a = (2*l+1)*factorial<ValueType>(l-m);
  ValueType b = 4*M_PI*factorial<ValueType>(l+m);
  ValueType C = cuda::std::sqrt(a/b);

  auto Ylm = C * Plm * exp(cuda::std::complex<ValueType>(0,1)*(m*phi));

  auto [ Xm, Ym, Zm ] = sph2cart(phi, ValueType(M_PI)/2 - theta, abs(real(Ylm)));

  // Output location
  auto X = make_tensor<ValueType>(Xm.Shape());
  auto Y = make_tensor<ValueType>(Ym.Shape());
  auto Z = make_tensor<ValueType>(Zm.Shape());

  (X = Xm, Y = Ym, Z=Zm).run(exec);

  exec.sync();

#if MATX_ENABLE_VIZ
  matx::viz::surf(X, Y, Z, "test-viz.html");
#endif
  CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
}
