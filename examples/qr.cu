////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "matx.h"
#include <cassert>
#include <cstdio>
#include <math.h>

using namespace matx;

int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();

  //using AType = double;
  using AType = cuda::std::complex<float>;
  
  hipStream_t stream = 0;
  cudaExecutor exec{stream};
  int batch = 1; 

  int m = 4;
  int n = 5;
 
  auto A = make_tensor<AType>({batch, m, n});
  auto QR = make_tensor<AType>({batch, m, n});
  auto QTQ = make_tensor<AType>({batch, m, m});
  auto Q = make_tensor<AType>({batch, m, m});
  auto R = make_tensor<AType>({batch, m, n});

  (A = random<float>(A.Shape(), NORMAL)).run(exec);

#if 0
  exec.sync();
  A(0,0,0) = 10000; A(0,0,1) = 10001;
  A(0,1,0) = 10001; A(0,1,1) = 10002;
  A(0,2,0) = 10002; A(0,2,1) = 10003;
  A(0,3,0) = 10003; A(0,3,1) = 10004;
  A(0,4,0) = 10004; A(0,4,1) = 10005;
#endif

  (mtie(Q, R) = qr(A)).run(exec);

  (QR = matmul(Q, R)).run(exec);
  (QTQ = matmul(conj(transpose_matrix(Q)), Q)).run(exec);
  exec.sync();
  
  printf("Q:\n"); print(Q);
  printf("R:\n"); print(R);
  printf("QTQ:\n"); print(QTQ);
  printf("QR:\n"); print(QR);
  printf("A:\n"); print(A);

  CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
}
