////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "simple_pipeline.h"

int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();
  index_t numChannels = 16;
  index_t numPulses = 128;
  index_t numSamples = 9000;
  index_t waveformLength = 1000;
  uint32_t iterations = 100;

  std::cout << "Iterations: " << iterations << std::endl;
  std::cout << "numChannels: " << numChannels << std::endl;
  std::cout << "numPulses: " << numPulses << std::endl;
  std::cout << "numNumSamples: " << numSamples << std::endl;
  std::cout << "waveformLength: " << waveformLength << std::endl;

  // cuda stream to place work in
  hipStream_t stream;
  hipStreamCreate(&stream);

  // create some events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  printf("Initializing data structures...\n");
  auto radar =
      RadarPipeline(numPulses, numSamples, waveformLength, numChannels, stream);
  radar.GetInputView()->PrefetchDevice(stream);

  printf("Running test...\n");
  hipStreamSynchronize(stream);
  hipEventRecord(start, stream);
  for (uint32_t i = 0; i < iterations; i++) {
    radar.PulseCompression();
    radar.ThreePulseCanceller();
    radar.DopplerProcessing();
    radar.CFARDetections();
  }

  hipEventRecord(stop, stream);
  hipStreamSynchronize(stream);
  float time_ms;
  hipEventElapsedTime(&time_ms, start, stop);
  float time_s = time_ms * .001f;

  printf("Pipeline rate: %f pulses/channel/sec\n",
         static_cast<float>(iterations * numPulses) / time_s);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);

  hipDeviceSynchronize();
  CUDA_CHECK_LAST_ERROR();

  matxPrintMemoryStatistics();

  printf("Done\n");

  MATX_EXIT_HANDLER();
  return 0;
}
