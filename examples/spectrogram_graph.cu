#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "matx.h"
#include <cassert>
#include <cstdio>
#include <math.h>
#include <memory>

using namespace matx;
#define FFT_TYPE HIPFFT_C2C

/** Create a spectrogram of a signal
 *
 * This example creates a set of data representing signal power versus frequency
 * and time. Traditionally the signal power is plotted as the Z dimension using
 * color, and time/frequency are the X/Y axes. The time taken to run the
 * spectrogram is computed, and a simple scatter plot is output. This version
 * does uses CUDA graphs, and records the workload on the second iteration of
 * the intialization loop. The first iteration is used only for plan caching and
 * should not include any graph recording.
 */

int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();

  using complex = cuda::std::complex<float>;
  hipGraph_t graph;
  hipGraphExec_t instance;

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float fs = 10000;
  index_t N = 100000;
  float amp = static_cast<float>(2 * sqrt(2));
  index_t nperseg = 256;
  index_t nfft = 256;
  index_t noverlap = nperseg / 8;
  index_t nstep = nperseg - noverlap;
  constexpr uint32_t num_iterations = 100;
  float time_ms;

  std::array<index_t, 1> num_samps{N};
  std::array<index_t, 1> half_win{nfft / 2 + 1};
  std::array<index_t, 1> s_time_shape{(N - noverlap) / nstep};

  tensor_t<float, 1> time({N});
  tensor_t<float, 1> modulation({N});
  tensor_t<float, 1> carrier({N});
  tensor_t<float, 1> noise({N});
  tensor_t<float, 1> x({N});
  auto freqs = make_tensor<float>(half_win);
  tensor_t<complex, 2> fftStackedMatrix(
      {(N - noverlap) / nstep, nfft / 2 + 1});
  tensor_t<float, 1> s_time({(N - noverlap) / nstep});

  randomGenerator_t<float> randData({N}, 0);
  auto randDataView = randData.GetTensorView<1>(num_samps, NORMAL);

  // Set up all static buffers
  // time = np.arange(N) / float(fs)
  (time = linspace<0>(num_samps, 0.0f, static_cast<float>(N) - 1.0f) / fs)
      .run(stream);
  // mod = 500 * np.cos(2*np.pi*0.25*time)
  (modulation = 500 * cos(2 * M_PI * 0.25 * time)).run(stream);
  // carrier = amp * np.sin(2*np.pi*3e3*time + modulation)
  (carrier = amp * sin(2 * M_PI * 3000 * time + modulation)).run(stream);
  // noise = 0.01 * fs / 2 * np.random.randn(time.shape)
  (noise = sqrt(0.01 * fs / 2) * randDataView).run(stream);
  // noise *= np.exp(-time/5)
  (noise = noise * exp(-1.0f * time / 5.0f)).run(stream);
  // x = carrier + noise
  (x = carrier + noise).run(stream);

  for (uint32_t i = 0; i < 2; i++) {
    // Record graph on second loop to get rid of plan caching in the graph
    if (i == 1) {
      hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    }

    // DFT Sample Frequencies (rfftfreq)
    (freqs = (1.0 / (static_cast<float>(nfft) * 1 / fs)) *
               linspace<0>(half_win, 0.0f, static_cast<float>(nfft) / 2.0f))
        .run(stream);

    // Create overlapping matrix of segments.
    auto stackedMatrix = x.OverlapView({nperseg}, {nstep});
    // FFT along rows
    fft(fftStackedMatrix, stackedMatrix, 0, stream);
    // Absolute value
    (fftStackedMatrix = conj(fftStackedMatrix) * fftStackedMatrix)
        .run(stream);
    // Get real part and transpose
    auto Sxx = fftStackedMatrix.RealView().Permute({1, 0});

    // Spectral time axis
    (s_time = linspace<0>(s_time_shape, static_cast<float>(nperseg) / 2.0f,
                           static_cast<float>(N - nperseg) / 2.0f + 1) /
                fs)
        .run(stream);

    if (i == 1) {
      hipStreamEndCapture(stream, &graph);
      hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

#if MATX_ENABLE_VIZ
      // Generate a spectrogram visualization using a contour plot
      viz::contour(time, freqs, Sxx);
#else
      printf("Not outputting plot since visualizations disabled\n");
#endif            
    }
  }

  // Time graph execution of same kernels
  hipEventRecord(start, stream);
  for (uint32_t i = 0; i < num_iterations; i++) {
    hipGraphLaunch(instance, stream);
  }
  hipEventRecord(stop, stream);
  hipStreamSynchronize(stream);
  hipEventElapsedTime(&time_ms, start, stop);

  printf("Spectrogram Time With Graphs = %.2fus per iteration\n",
         time_ms * 1e3 / num_iterations);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);
  CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
}
