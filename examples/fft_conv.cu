#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "matx.h"
#include <cassert>
#include <cstdio>
#include <cuda/std/ccomplex>

using namespace matx;

/**
 * FFT Convolution
 *
 * This example shows how to perform an FFT convolution using the MatX library.
 * The example shows the convolution theorem of:
 *
 * \f(h*x \leftrightarrow H \cdot X$  \f)
 *
 * Namely, a convolution in the time domain is a point-wise multiplication in
 * the frequency domain. In this example we start with two signals in the time
 * domain, convert them to frequency domain, perform the multiply, then convert
 * them back to the time domain. This should give very close results to
 * performing a direct convolution in the time domain, so the results are
 * compared to a direct convolution. They will not match identically since the
 * types and order of operations are different, but they will match within a
 * close margin.
 *
 * FFT convolution is frequently used in signal processing when a signal or
 * filter is larger than a threshold, since it will outperform direct
 * convolution past this threshold. Another benefit of FFT convolution is the
 * number of operations is the same, regardless of the filter size. This allows
 * a user to FFT a very long filter one time, and that buffer can be used many
 * times for any incoming samples.
 *
 * For smaller signal sizes, the FFT convolution typically performs worse since
 * there is some buffer and 3 FFT operations (2 for FFT of signal and filter,
 * and 1 IFFT after the multiply) that causes the setup time to dominate.
 *
 */
int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();
  using complex = cuda::std::complex<float>;

  index_t signal_size = 1ULL << 16;
  index_t filter_size = 16;
  index_t batches = 8;
  index_t filtered_size = signal_size + filter_size - 1;

  // Create time domain buffers
  auto sig_time  = make_tensor<complex>({batches, signal_size});
  auto filt_time = make_tensor<complex>({batches, filter_size});
  auto time_out  = make_tensor<complex>({batches, filtered_size});

  // Frequency domain buffers
  auto sig_freq  = make_tensor<complex>({batches, filtered_size});
  auto filt_freq = make_tensor<complex>({batches, filtered_size});

  for (index_t b = 0; b < batches; b++) {
    // Fill the time domain signals with data
    for (index_t i = 0; i < signal_size; i++) {
      sig_time(b,i) = {-1.0f * (2.0f * static_cast<float>(i % 2) + 1.0f) *
                            (static_cast<float>(i % 10) / 10.0f) +
                        0.1f,
                    -1.0f * (static_cast<float>(i % 2) == 0.0f) *
                            (static_cast<float>(i % 10) / 5.0f) -
                        0.1f};
    }
    for (index_t i = 0; i < filter_size; i++) {
      filt_time(b,i) = {static_cast<float>(i) / static_cast<float>(filter_size),
                      static_cast<float>(-i) / static_cast<float>(filter_size) +
                          0.5f};
    }
  }

  // Prefetch the data we just created
  sig_time.PrefetchDevice(0);
  filt_time.PrefetchDevice(0);

  // Perform the FFT in-place on both signal and filter
  fft(sig_freq, sig_time);
  fft(filt_freq, filt_time);

  // Perform the pointwise multiply. Overwrite signal buffer with result
  (sig_freq = sig_freq * filt_freq).run();

  // IFFT in-place
  ifft(sig_freq, sig_freq);

  // Now the sig_freq view contains the full convolution result. Verify against
  // a direct convolution. The conv1d function only accepts a 1D filter, so we
  // create a sliced view here.
  auto filt1 = filt_time.Slice<1>({0,0}, {matxDropDim, matxEnd});
  conv1d(time_out, sig_time, filt1, matxConvCorrMode_t::MATX_C_MODE_FULL,
          0);

  hipStreamSynchronize(0);
 
  // Compare signals
  for (index_t b = 0; b < batches; b++) {
    for (index_t i = 0; i < filtered_size; i++) {
      if (fabs(time_out(b,i).real() - sig_freq(b,i).real()) > 0.001 ||
          fabs(time_out(b,i).imag() - sig_freq(b,i).imag()) > 0.001) {
        std::cout <<
            "Verification failed at item " << i << ". Direct=" << time_out(b,i).real() << " " << time_out(b,i).imag() << ", FFT=" <<
            sig_freq(b,i).real() << " " <<
            sig_freq(b,i).imag() << "\n";
        return -1;
      }
    }
  }

  std::cout << "Verification successful" << std::endl;

  CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
}
