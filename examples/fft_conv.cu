#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "matx.h"
#include <cassert>
#include <cstdio>
#include <cuda/std/ccomplex>

using namespace matx;

/**
 * FFT Convolution
 *
 * This example shows how to perform an FFT convolution using the MatX library.
 * The example shows the convolution theorem of:
 *
 * \f(h*x \leftrightarrow H \cdot X$  \f)
 *
 * Namely, a convolution in the time domain is a point-wise multiplication in
 * the frequency domain. In this example we start with two signals in the time
 * domain, convert them to frequency domain, perform the multiply, then convert
 * them back to the time domain. This should give very close results to
 * performing a direct convolution in the time domain, so the results are
 * compared to a direct convolution. They will not match identically since the
 * types and order of operations are different, but they will match within a
 * close margin.
 *
 * FFT convolution is frequently used in signal processing when a signal or
 * filter is larger than a threshold, since it will outperform direct
 * convolution past this threshold. Another benefit of FFT convolution is the
 * number of operations is the same, regardless of the filter size. This allows
 * a user to FFT a very long filter one time, and that buffer can be used many
 * times for any incoming samples.
 *
 * For smaller signal sizes, the FFT convolution typically performs worse since
 * there is some buffer and 3 FFT operations (2 for FFT of signal and filter,
 * and 1 IFFT after the multiply) that causes the setup time to dominate.
 * 
 * Note that the conv1d() operator has a mode to perform FFT-based convolution
 * automatically.
 *
 */
int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();
  using complex = cuda::std::complex<float>;
  cudaExecutor exec{};

  index_t signal_size = 1ULL << 16;
  index_t filter_size = 16;
  index_t batches = 8;
  index_t filtered_size = signal_size + filter_size - 1;
  float separate_ms;
  float fused_ms;
  constexpr int iterations = 100;
  hipStream_t stream;
  hipStreamCreate(&stream);  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);  

  // Create time domain buffers
  auto sig_time  = make_tensor<complex>({batches, signal_size});
  auto filt_time = make_tensor<complex>({batches, filter_size});
  auto time_out  = make_tensor<complex>({batches, filtered_size});

  // Frequency domain buffers
  auto sig_freq  = make_tensor<complex>({batches, filtered_size});
  auto filt_freq = make_tensor<complex>({batches, filtered_size});

  for (index_t b = 0; b < batches; b++) {
    // Fill the time domain signals with data
    for (index_t i = 0; i < signal_size; i++) {
      sig_time(b,i) = {-1.0f * (2.0f * static_cast<float>(i % 2) + 1.0f) *
                            (static_cast<float>(i % 10) / 10.0f) +
                        0.1f,
                    -1.0f * (static_cast<float>(i % 2) == 0.0f) *
                            (static_cast<float>(i % 10) / 5.0f) -
                        0.1f};
    }
    for (index_t i = 0; i < filter_size; i++) {
      filt_time(b,i) = {static_cast<float>(i) / static_cast<float>(filter_size),
                      static_cast<float>(-i) / static_cast<float>(filter_size) +
                          0.5f};
    }
  }

  // Perform the FFT in-place on both signal and filter
  for (int i = 0; i < iterations; i++) {
    if (i == 1) {
      hipEventRecord(start, stream);
    }    
    (sig_freq = fft(sig_time, filtered_size)).run(exec);
    (filt_freq = fft(filt_time, filtered_size)).run(exec);

    (sig_freq = sig_freq * filt_freq).run(exec);

    // IFFT in-place
    (sig_freq = ifft(sig_freq)).run(exec);
    
  }

  hipEventRecord(stop, stream);
  exec.sync();
  hipEventElapsedTime(&separate_ms, start, stop);   

  for (int i = 0; i < iterations; i++) {
    if (i == 1) {
      hipEventRecord(start, stream);
    }
    (sig_freq = ifft(fft(sig_time, filtered_size) * fft(filt_time, filtered_size))).run(exec);
  }
  
  hipEventRecord(stop, stream);
  exec.sync();
  hipEventElapsedTime(&fused_ms, start, stop);  

  printf("FFT runtimes for separate = %.2f ms, fused = %.2f ms\n", separate_ms/(iterations-1), fused_ms/(iterations-1));

  // Now the sig_freq view contains the full convolution result. Verify against
  // a direct convolution. The conv1d function only accepts a 1D filter, so we
  // create a sliced view here.
  auto filt1 = filt_time.Slice<1>({0,0}, {matxDropDim, matxEnd});
  (time_out = conv1d(sig_time, filt1, matxConvCorrMode_t::MATX_C_MODE_FULL)).run(exec);

  exec.sync();
 
  // Compare signals
  for (index_t b = 0; b < batches; b++) {
    for (index_t i = 0; i < filtered_size; i++) {
      if (fabs(time_out(b,i).real() - sig_freq(b,i).real()) > 0.001 ||
          fabs(time_out(b,i).imag() - sig_freq(b,i).imag()) > 0.001) {
        std::cout <<
            "Verification failed at item " << i << ". Direct=" << time_out(b,i).real() << " " << time_out(b,i).imag() << ", FFT=" <<
            sig_freq(b,i).real() << " " <<
            sig_freq(b,i).imag() << "\n";
        return -1;
      }
    }
  }

  std::cout << "Verification successful" << std::endl;

  CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
}