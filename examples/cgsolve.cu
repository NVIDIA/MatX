#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include <cassert>
#include <cstdio>
#include <math.h>
#include <memory>

#include "matx.h"

using namespace matx;
int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  using TypeParam = double;
  MATX_ENTER_HANDLER();

  int max_iters=100;
  int N = 120;
  int BATCH = 900;

  auto A = make_tensor<TypeParam, 3> ({BATCH, N, N});
  auto X = make_tensor<TypeParam, 2> ({BATCH, N});
  auto B = make_tensor<TypeParam, 2> ({BATCH, N});
  auto Bout = make_tensor<TypeParam, 2> ({BATCH, N});
  auto norm = make_tensor<TypeParam, 1>({BATCH});
  auto maxn = make_tensor<TypeParam>({});

  cudaExecutor exec{};

  // Simple Poisson matrix
  for(int b = 0; b < BATCH; b++) {
    for(int i = 0; i < N; i++) {
      B(b,i) = TypeParam(1+b);

      for(int j = 0; j < N; j++) {
        if(i==j)
          A(b,i,j) = 2;
        else if( i == j-1)
          A(b,i,j) = -1;
        else if (i == j+1)
          A(b,i,j) = -1;
        else
          A(b,i,j) = 0;
      }
    }
  }

  (X = TypeParam(1)).run(exec);

  (X = cgsolve(A, B, .0001, max_iters)).run(exec);
  // example-begin sync-test-1
  (Bout = matvec(A, X)).run(exec);
  (norm = sum((Bout-B)*(Bout-B))).run(exec);
  (maxn = matx::max(sqrt(norm))).run(exec);

  exec.sync();
  // example-end sync-test-1
  printf ("max l2 norm: %f\n", (float)sqrt(maxn()));

  CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
}
